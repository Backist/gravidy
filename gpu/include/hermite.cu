#include "hip/hip_runtime.h"
#include "hermite.cuh"

void integrate_gpu()
{
    double ATIME = 1.0e+10; // Actual integration time
    double ITIME = 0.0;     // Integration time
    int nact     = 0;       // Active particles
    int nsteps   = 0;       // Amount of steps per particles on the system
    iterations   = 0;       // Iterations of the integration

    // Tmp setting nblocks and nthreads
    nthreads = BSIZE;
    nblocks = ceil(n/(float)nthreads);

    // Copying the input file information from the CPU to the GPU
    CUDA_SAFE_CALL(hipMemcpy(d_r,  h_r,  d4_size,hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_v,  h_v,  d4_size,hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_m,  h_m,  f1_size,hipMemcpyHostToDevice));

    gpu_init_acc_jrk();   // Initial calculation of a and a1
    //init_acc_jrk();   // Initial calculation of a and a1

    // Copying a and a1 from the GPU to the CPU
    CUDA_SAFE_CALL(hipMemcpy(h_f,  d_f,  sizeof(Forces) * n ,hipMemcpyDeviceToHost));

    init_dt(&ATIME);  // Initial calculation of time-steps using simple equation
    //init_dt2(&ATIME); // Initial calculation of time-steps using complete equation

    energy_ini = gpu_energy(); // Initial calculation of the energy of the system
    energy_tmp = energy_ini;  // Saving initial energy, to calculate errors

    get_energy_log(ITIME, iterations, nsteps, out, energy_tmp); // First log of the integration

    float tmp_time = 0.0f;
    gpu_time = 0.0f;

    while (ITIME < int_time)
    {
        ITIME = ATIME;                         // New integration time
        nact = find_particles_to_move(ITIME);  // Find particles to move (nact)
        save_old(nact);                        // Save old information

        if (nact < n * alpha)
        {
            predicted_pos_vel(ITIME);
            update_acc_jrk(nact);
            correction_pos_vel(ITIME, nact);       // Correct r and v of nact particles
            cpu_iterations++;
        }
        else
        {
            tmp_time = (float)clock()/CLOCKS_PER_SEC;
            //CUDA_SAFE_CALL(hipMemcpy(d_move, h_move, i1_size, hipMemcpyHostToDevice));
            //CUDA_SAFE_CALL(hipMemcpy(d_t,  h_t,  d1_size, hipMemcpyHostToDevice));
            //gpu_predicted_pos_vel(ITIME);
            predicted_pos_vel(ITIME);
            CUDA_SAFE_CALL(hipMemcpy(d_p, h_p, sizeof(Predictor) * n,hipMemcpyHostToDevice));
            gpu_update(nact);     // Update a and a1 of nact particles
            //CUDA_SAFE_CALL(hipMemcpy(h_f,  d_f,  sizeof(Forces) * n ,hipMemcpyDeviceToHost));
            correction_pos_vel(ITIME, nact);       // Correct r and v of nact particles
            CUDA_SAFE_CALL(hipMemcpy(d_r, h_r, d4_size, hipMemcpyHostToDevice));
            CUDA_SAFE_CALL(hipMemcpy(d_v, h_v, d4_size, hipMemcpyHostToDevice));
            gpu_time += (float)clock()/CLOCKS_PER_SEC - tmp_time;
            gpu_iterations++;
        }

        next_itime(&ATIME);                    // Find next integration time

        if(std::ceil(ITIME) == ITIME)          // Print log in every integer ITIME
        {
           get_energy_log(ITIME, iterations, nsteps, out, gpu_energy());
        }

        nsteps += nact;                        // Update nsteps with nact
        iterations++;                          // Increase iterations
    }
}
