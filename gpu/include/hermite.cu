#include "hip/hip_runtime.h"
#include "hermite.cuh"

void integrate_gpu()
{
    double ATIME = 1.0e+10; // Actual integration time
    double ITIME = 0.0;     // Integration time
    int nact     = 0;       // Active particles
    int nsteps   = 0;       // Amount of steps per particles on the system
    iterations   = 0;       // Iterations of the integration

    // Tmp setting nblocks and nthreads
    nthreads = BSIZE;
    nblocks = ceil(n/(float)nthreads);

    // Copying the input file information from the CPU to the GPU
    CUDA_SAFE_CALL(hipMemcpy(d_r,  h_r,  d4_size,hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_v,  h_v,  d4_size,hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_m,  h_m,  f1_size,hipMemcpyHostToDevice));

    gpu_init_acc_jrk();   // Initial calculation of a and a1

    // Copying a and a1 from the GPU to the CPU
    CUDA_SAFE_CALL(hipMemcpy(h_f,  d_f,  sizeof(Forces) * n ,hipMemcpyDeviceToHost));

    init_dt(&ATIME);  // Initial calculation of time-steps using simple equation

    energy_ini = gpu_energy(); // Initial calculation of the energy of the system
    energy_tmp = energy_ini;  // Saving initial energy, to calculate errors

    get_energy_log(ITIME, iterations, nsteps, out, energy_tmp); // First log of the integration

    float tmp_time = 0.0f;
    gpu_time = 0.0f;


    float kernel_time_tmp = 0.0f;
    float kernel_time = 0.0f;
    int interactions = 0;

    while (ITIME < int_time)
    {
        ITIME = ATIME;                         // New integration time
        nact = find_particles_to_move(ITIME);  // Find particles to move (nact)
        save_old(nact);                        // Save old information

        if (nact < beta)
        {
            // Predicted r and v
            predicted_pos_vel(ITIME);
            // Update the forces of the nact particles
            update_acc_jrk(nact);
            // Correct r and v
            correction_pos_vel(ITIME, nact);
            cpu_iterations++;
        }
        else
        {
            tmp_time = (float)clock()/CLOCKS_PER_SEC;

            predicted_pos_vel(ITIME);

            kernel_time_tmp = (float)clock()/CLOCKS_PER_SEC;
            gpu_update(nact);     // Update a and a1 of nact particles
            kernel_time += (float)clock()/CLOCKS_PER_SEC - kernel_time_tmp;

            correction_pos_vel(ITIME, nact);       // Correct r and v of nact particles

            CUDA_SAFE_CALL(hipMemcpy(d_r, h_r, d4_size, hipMemcpyHostToDevice));
            CUDA_SAFE_CALL(hipMemcpy(d_v, h_v, d4_size, hipMemcpyHostToDevice));

            gpu_time += (float)clock()/CLOCKS_PER_SEC - tmp_time;
            gpu_iterations++;
            interactions += nact * n;
        }

        next_itime(&ATIME);                    // Find next integration time

        //if(std::ceil(ITIME) == ITIME)          // Print log in every integer ITIME
        if(nact == n)          // Print log in every integer ITIME
        {
           get_energy_log(ITIME, iterations, nsteps, out, gpu_energy());
        }

        nsteps += nact;                        // Update nsteps with nact
        iterations++;                          // Increase iterations
        printf("%d\n", nact);
    }
    //printf("GG: %f\n", 60.10e-9 * (interactions / kernel_time));
}
