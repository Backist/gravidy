#include "memory_gpu.cuh"

/*
 * @fn alloc_vectors_gpu()
 *
 */
void alloc_vectors_gpu()
{
    /*
     * GPU pointers
     */
    CUDA_SAFE_CALL(hipMalloc((void**)&d_r,     d4_size));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_v,     d4_size));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_f,     sizeof(Forces) * n));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_p,     sizeof(Predictor) * n));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_ekin,  d1_size));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_epot,  d1_size));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_t,     d1_size));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_dt,    d1_size));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_m,     f1_size));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_move,  i1_size));

    CUDA_SAFE_CALL(hipMalloc((void**)&d_i,    sizeof(Predictor) * n));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_fout, sizeof(Forces) * n * NJBLOCK));
    CUDA_SAFE_CALL(hipMemset(d_i,  0,sizeof(Predictor) * n));
    CUDA_SAFE_CALL(hipMemset(d_fout,  0,sizeof(Forces) * n * NJBLOCK));

    /*
     * Memset
     */
    CUDA_SAFE_CALL(hipMemset(d_r, 0, d4_size));
    CUDA_SAFE_CALL(hipMemset(d_v, 0, d4_size));
    CUDA_SAFE_CALL(hipMemset(d_f, 0, sizeof(Forces) * n));
    CUDA_SAFE_CALL(hipMemset(d_p, 0, sizeof(Predictor) * n));
    CUDA_SAFE_CALL(hipMemset(d_ekin,  0,d1_size));
    CUDA_SAFE_CALL(hipMemset(d_epot,  0,d1_size));
    CUDA_SAFE_CALL(hipMemset(d_t,     0,d1_size));
    CUDA_SAFE_CALL(hipMemset(d_dt,    0,d1_size));
    CUDA_SAFE_CALL(hipMemset(d_m,     0,f1_size));
    CUDA_SAFE_CALL(hipMemset(d_move,  0,i1_size));
}


/*
 * @fn free_vectors_gpu()
 *
 * @brief
 *  Free memory on the GPU
 */
void free_vectors_gpu()
{
    CUDA_SAFE_CALL(hipFree(d_r));
    CUDA_SAFE_CALL(hipFree(d_v));
    CUDA_SAFE_CALL(hipFree(d_m));
    CUDA_SAFE_CALL(hipFree(d_t));
    CUDA_SAFE_CALL(hipFree(d_f));
    CUDA_SAFE_CALL(hipFree(d_p));
    CUDA_SAFE_CALL(hipFree(d_dt));
    CUDA_SAFE_CALL(hipFree(d_ekin));
    CUDA_SAFE_CALL(hipFree(d_epot));
    CUDA_SAFE_CALL(hipFree(d_move));

    CUDA_SAFE_CALL(hipFree(d_i));
    CUDA_SAFE_CALL(hipFree(d_fout));
}
