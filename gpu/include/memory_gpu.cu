#include "memory_gpu.cuh"

/*
 * @fn alloc_vectors_gpu()
 *
 */
void alloc_vectors_gpu()
{
    /*
     * GPU pointers
     */
    size_t fsize = sizeof(Forces) * n;
    size_t psize = sizeof(Predictor) * n;

    CUDA_SAFE_CALL(hipMalloc((void**)&d_r,     d4_size));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_v,     d4_size));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_f,     fsize));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_p,     psize));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_ekin,  d1_size));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_epot,  d1_size));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_t,     d1_size));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_dt,    d1_size));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_m,     f1_size));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_move,  i1_size));

    CUDA_SAFE_CALL(hipMalloc((void**)&d_i,    psize));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_fout, fsize * NJBLOCK));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_fout_tmp, fsize * NJBLOCK));
    CUDA_SAFE_CALL(hipMemset(d_i,  0, psize));
    CUDA_SAFE_CALL(hipMemset(d_fout,  0, fsize * NJBLOCK));
    CUDA_SAFE_CALL(hipMemset(d_fout_tmp,  0, fsize * NJBLOCK));

    /*
     * Memset
     */
    CUDA_SAFE_CALL(hipMemset(d_r, 0, d4_size));
    CUDA_SAFE_CALL(hipMemset(d_v, 0, d4_size));
    CUDA_SAFE_CALL(hipMemset(d_f, 0, fsize));
    CUDA_SAFE_CALL(hipMemset(d_p, 0, psize));
    CUDA_SAFE_CALL(hipMemset(d_ekin,  0,d1_size));
    CUDA_SAFE_CALL(hipMemset(d_epot,  0,d1_size));
    CUDA_SAFE_CALL(hipMemset(d_t,     0,d1_size));
    CUDA_SAFE_CALL(hipMemset(d_dt,    0,d1_size));
    CUDA_SAFE_CALL(hipMemset(d_m,     0,f1_size));
    CUDA_SAFE_CALL(hipMemset(d_move,  0,i1_size));
}


/*
 * @fn free_vectors_gpu()
 *
 * @brief
 *  Free memory on the GPU
 */
void free_vectors_gpu()
{
    CUDA_SAFE_CALL(hipFree(d_r));
    CUDA_SAFE_CALL(hipFree(d_v));
    CUDA_SAFE_CALL(hipFree(d_m));
    CUDA_SAFE_CALL(hipFree(d_t));
    CUDA_SAFE_CALL(hipFree(d_f));
    CUDA_SAFE_CALL(hipFree(d_p));
    CUDA_SAFE_CALL(hipFree(d_dt));
    CUDA_SAFE_CALL(hipFree(d_ekin));
    CUDA_SAFE_CALL(hipFree(d_epot));
    CUDA_SAFE_CALL(hipFree(d_move));

    CUDA_SAFE_CALL(hipFree(d_i));
    CUDA_SAFE_CALL(hipFree(d_fout));
}
