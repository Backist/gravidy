#include "hip/hip_runtime.h"
#include "dynamics_gpu_kernels.cuh"

__global__ void k_energy(double4 *r,
                         double4 *v,
                         double *ekin,
                         double *epot,
                         float *m,
                         int n)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j;
    double ekin_tmp = 0.0;
    double epot_tmp = 0.0;

    if (i < n)
    {
        epot_tmp = 0.0;
        for (j = i+1; j < n; j++)
        {
            double rx = r[j].x - r[i].x;
            double ry = r[j].y - r[i].y;
            double rz = r[j].z - r[i].z;
            double r2 = rx*rx + ry*ry + rz*rz;
            epot_tmp -= (m[i] * m[j]) * rsqrt(r2);
        }

        double vx = v[i].x * v[i].x;
        double vy = v[i].y * v[i].y;
        double vz = v[i].z * v[i].z;
        double v2 = vx + vy + vz;
        ekin_tmp = 0.5 * m[i] * v2;

        ekin[i] = ekin_tmp;
        epot[i] = epot_tmp;
    }
}

__global__ void k_init_acc_jrk (double4 *r,
                                double4 *v,
                                Forces *d_f,
                                float *m,
                                int n,
                                double e2)
{

    extern __shared__ double4 sh[];
    double4 *sr = (double4*)sh;
    double4 *sv = (double4*)&sr[blockDim.x];

    Forces ff;
    ff.a[0] = 0.0;
    ff.a[1] = 0.0;
    ff.a[2] = 0.0;
    ff.a1[0] = 0.0;
    ff.a1[1] = 0.0;
    ff.a1[2] = 0.0;
    float mj;

    int id = threadIdx.x + blockDim.x * blockIdx.x;
    int tx = threadIdx.x;

    if (id < n)
    {
        double4 pos = r[id];
        double4 vel = v[id];

        int tile = 0;
        for (int i = 0; i < n; i += BSIZE)
        {
            int idx = tile * BSIZE + tx;

            sr[tx]   = r[idx];
            sv[tx]   = v[idx];
            mj = m[idx];
            __syncthreads();

            for (int k = 0; k < BSIZE; k++)
            {
                k_force_calculation(pos, vel, sr[k], sv[k], ff, mj,e2);
            }
            __syncthreads();
            tile++;
        }

        d_f[id] = ff;
    }
}

/*
 * @fn k_predicted_pos_vel
 *
 * @param to do
 *
 * @desc GPU Kernel to calculate the predicted position and velocity
 *       of all the particles.
 *
 * @note Working properly.
 */
__global__ void k_predicted_pos_vel(double4 *d_r,
                                    double4 *d_v,
                                    Forces *d_f,
                                    Predictor *d_p,
                                    double *d_t,
                                    double ITIME,
                                    int n)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    if (i < n)
    {
        double dt = ITIME - d_t[i];
        double dt2 = (dt  * dt)/2;
        double dt3 = (dt2 * dt)/6;

        d_p[i].r[0] = (dt3/6 * d_f[i].a1[0]) + (dt2/2 * d_f[i].a[0]) + (dt * d_v[i].x) + d_r[i].x;
        d_p[i].r[1] = (dt3/6 * d_f[i].a1[1]) + (dt2/2 * d_f[i].a[1]) + (dt * d_v[i].y) + d_r[i].y;
        d_p[i].r[2] = (dt3/6 * d_f[i].a1[2]) + (dt2/2 * d_f[i].a[2]) + (dt * d_v[i].z) + d_r[i].z;

        d_p[i].v[0] = (dt2/2 * d_f[i].a1[0]) + (dt * d_f[i].a[0]) + d_v[i].x;
        d_p[i].v[1] = (dt2/2 * d_f[i].a1[1]) + (dt * d_f[i].a[1]) + d_v[i].y;
        d_p[i].v[2] = (dt2/2 * d_f[i].a1[2]) + (dt * d_f[i].a[2]) + d_v[i].z;
    }
}

/*
 * @fn k_force_calculation
 *
 * @desc GPU Kernel which calculates the interaction between
 *       a i-particle and a j-particle.
 *
 * @note Working properly.
 *
 */
__device__ void k_force_calculation(double4 i_pos, double4 i_vel,
                                    double4 j_pos, double4 j_vel,
                                    Forces &d_f,
                                    float   j_mass,
                                    double e2)
{
    double rx = j_pos.x - i_pos.x;
    double ry = j_pos.y - i_pos.y;
    double rz = j_pos.z - i_pos.z;

    double vx = j_vel.x - i_vel.x;
    double vy = j_vel.y - i_vel.y;
    double vz = j_vel.z - i_vel.z;

    double r2 = rx*rx + ry*ry + rz*rz + e2;
    double rinv = rsqrt(r2);
    double r2inv = rinv  * rinv;
    double r3inv = r2inv * rinv;
    double r5inv = r2inv * r3inv;
    double mr3inv = r3inv * j_mass;
    double mr5inv = r5inv * j_mass;

    double rv = rx*vx + ry*vy + rz*vz;

    d_f.a[0] += (rx * mr3inv);
    d_f.a[1] += (ry * mr3inv);
    d_f.a[2] += (rz * mr3inv);

    d_f.a1[0] += (vx * mr3inv - (3 * rv) * rx * mr5inv);
    d_f.a1[1] += (vy * mr3inv - (3 * rv) * ry * mr5inv);
    d_f.a1[2] += (vz * mr3inv - (3 * rv) * rz * mr5inv);
}

__device__ void k_force_calculation2(Predictor i_p,
                                     Predictor j_p,
                                     Forces &d_f,
                                     float   j_mass,
                                     double e2)
{
    double rx = j_p.r[0] - i_p.r[0];
    double ry = j_p.r[1] - i_p.r[1];
    double rz = j_p.r[2] - i_p.r[2];

    double vx = j_p.v[0] - i_p.v[0];
    double vy = j_p.v[1] - i_p.v[1];
    double vz = j_p.v[2] - i_p.v[2];

    double r2 = rx*rx + ry*ry + rz*rz + e2;
    double rinv = rsqrt(r2);
    double r2inv = rinv  * rinv;
    double r3inv = r2inv * rinv;
    double r5inv = r2inv * r3inv;
    double mr3inv = r3inv * j_mass;
    double mr5inv = r5inv * j_mass;

    double rv = rx*vx + ry*vy + rz*vz;

    d_f.a[0] += (rx * mr3inv);
    d_f.a[1] += (ry * mr3inv);
    d_f.a[2] += (rz * mr3inv);

    d_f.a1[0] += (vx * mr3inv - (3 * rv) * rx * mr5inv);
    d_f.a1[1] += (vy * mr3inv - (3 * rv) * ry * mr5inv);
    d_f.a1[2] += (vz * mr3inv - (3 * rv) * rz * mr5inv);
}

__global__ void k_update(Predictor *d_i,
                         Predictor *d_j,
                         Forces *d_fout,
                         float *d_m,
                         int n,
                         int total,
                         double e2)
{
    int ibid = blockIdx.x;
    int jbid = blockIdx.y;
    int tid = threadIdx.x;
    int iaddr = tid + blockDim.x * ibid;
    int jstart = (n * (jbid  )) / NJBLOCK;
    int jend   = (n * (jbid+1)) / NJBLOCK;
    float mj;

    Predictor ip = d_i[iaddr];
    Forces fo;
    fo.a[0] = 0.0;
    fo.a[1] = 0.0;
    fo.a[2] = 0.0;
    fo.a1[0] = 0.0;
    fo.a1[1] = 0.0;
    fo.a1[2] = 0.0;

    for(int j=jstart; j<jend; j+=BSIZE){
        __shared__ Predictor jpshare[BSIZE];
        __syncthreads();
        Predictor *src = (Predictor *)&d_j[j];
        Predictor *dst = (Predictor *)jpshare;
        dst[      tid] = src[      tid];
        dst[BSIZE+tid] = src[BSIZE+tid];
        mj = d_m[BSIZE + tid];
        __syncthreads();

        if(jend-j < BSIZE){
            for(int jj=0; jj<jend-j; jj++){
                Predictor jp = jpshare[jj];
                k_force_calculation2(ip, jp, fo, mj, e2);
            }
        }
        else{
            for(int jj=0; jj<BSIZE; jj++){
                Predictor jp = jpshare[jj];
                k_force_calculation2(ip, jp, fo, mj, e2);
            }
        }
    }
    d_fout[iaddr*NJBLOCK + jbid] = fo;

}

__global__ void reduce(Forces *d_in,
                       Forces *d_out,
                       unsigned int total)
{
    extern __shared__ Forces sdata[];

    const int xid   = threadIdx.x;
    const int bid   = blockIdx.x;
    const int iaddr = xid + blockDim.x * bid;

    sdata[xid] = d_in[iaddr];
    __syncthreads();

    if(xid < 8) sdata[xid] += sdata[xid + 8];
    if(xid < 4) sdata[xid] += sdata[xid + 4];
    if(xid < 2) sdata[xid] += sdata[xid + 2];
    if(xid < 1) sdata[xid] += sdata[xid + 1];

    if(xid == 0){
        d_out[bid] = sdata[0];
    }
}
