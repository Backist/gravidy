#include "hip/hip_runtime.h"
#include "dynamics_gpu_kernels.cuh"

__global__ void
k_energy(double4 *r, double4 *v, double *ekin, double *epot, float *m, int n)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j;
    double ekin_tmp = 0.0;
    double epot_tmp = 0.0;

    if (i < n)
    {
        epot_tmp = 0.0;
        for (j = i+1; j < n; j++)
        {
            double rx = r[j].x - r[i].x;
            double ry = r[j].y - r[i].y;
            double rz = r[j].z - r[i].z;
            double r2 = rx*rx + ry*ry + rz*rz;
            epot_tmp -= (m[i] * m[j]) * rsqrt(r2);
        }

        double vx = v[i].x * v[i].x;
        double vy = v[i].y * v[i].y;
        double vz = v[i].z * v[i].z;
        double v2 = vx + vy + vz;
        ekin_tmp = 0.5 * m[i] * v2;

        ekin[i] = ekin_tmp;
        epot[i] = epot_tmp;
    }
}

__global__ void k_update_acc_jrk_simple
(Predictor *d_p, Forces *d_f, float *m, int *move, int n, int total)
{
    extern __shared__ Predictor sh2[];
    Predictor *sp = (Predictor*)sh2;

    Forces ff;
    ff.a[0] = 0.0;
    ff.a[1] = 0.0;
    ff.a[2] = 0.0;
    ff.a1[0] = 0.0;
    ff.a1[1] = 0.0;
    ff.a1[2] = 0.0;

    float mj;

    Predictor p;

    int id = threadIdx.x + blockDim.x * blockIdx.x;
    int tx = threadIdx.x;

    int id_move = move[id];

    if(id_move != -1)
    {
        p = d_p[id_move];
    }

    int tile = 0;
    for (int i = 0; i < n; i += BSIZE)
    {
        int idx = tile * BSIZE + tx;

        sp[tx]   = d_p[idx];
        mj = m[idx];
        __syncthreads();

        for (int k = 0; k < BSIZE; k++)
        {
            if(id_move != -1)
            {
                k_force_calculation2(p, sp[k], ff, mj);
            }
        }
        __syncthreads();
        tile++;
    }

    if(id_move != -1)
    {
        d_f[id_move] = ff;
    }
}

__global__ void k_init_acc_jrk
(double4 *r, double4 *v, Forces *d_f, float *m, int n)
{

    extern __shared__ double4 sh[];
    double4 *sr = (double4*)sh;
    double4 *sv = (double4*)&sr[blockDim.x];

    Forces ff;
    ff.a[0] = 0.0;
    ff.a[1] = 0.0;
    ff.a[2] = 0.0;
    ff.a1[0] = 0.0;
    ff.a1[1] = 0.0;
    ff.a1[2] = 0.0;
    float mj;

    int id = threadIdx.x + blockDim.x * blockIdx.x;
    int tx = threadIdx.x;

    if (id < n)
    {
        double4 pos = r[id];
        double4 vel = v[id];

        int tile = 0;
        for (int i = 0; i < n; i += BSIZE)
        {
            int idx = tile * BSIZE + tx;

            sr[tx]   = r[idx];
            sv[tx]   = v[idx];
            mj = m[idx];
            __syncthreads();

            for (int k = 0; k < BSIZE; k++)
            {
                k_force_calculation(pos, vel, sr[k], sv[k], ff, mj);
            }
            __syncthreads();
            tile++;
        }

        d_f[id] = ff;
    }
}

/*
 * @fn k_force_calculation
 *
 * @desc GPU Kernel which calculates the interaction between
 *       a i-particle and a j-particle.
 *
 * @note Working properly.
 *
 */
__device__ void k_force_calculation(double4 i_pos, double4 i_vel,
                                    double4 j_pos, double4 j_vel,
                                    Forces &d_f,
                                    float   j_mass)
{
    double rx = j_pos.x - i_pos.x;
    double ry = j_pos.y - i_pos.y;
    double rz = j_pos.z - i_pos.z;

    double vx = j_vel.x - i_vel.x;
    double vy = j_vel.y - i_vel.y;
    double vz = j_vel.z - i_vel.z;

    double r2 = rx*rx + ry*ry + rz*rz + E2;
    double rinv = rsqrt(r2);
    double r2inv = rinv  * rinv;
    double r3inv = r2inv * rinv;
    double r5inv = r2inv * r3inv;
    double mr3inv = r3inv * j_mass;
    double mr5inv = r5inv * j_mass;

    double rv = rx*vx + ry*vy + rz*vz;

    d_f.a[0] += (rx * mr3inv);
    d_f.a[1] += (ry * mr3inv);
    d_f.a[2] += (rz * mr3inv);

    d_f.a1[0] += (vx * mr3inv - (3 * rv) * rx * mr5inv);
    d_f.a1[1] += (vy * mr3inv - (3 * rv) * ry * mr5inv);
    d_f.a1[2] += (vz * mr3inv - (3 * rv) * rz * mr5inv);
}

__device__ void k_force_calculation2(Predictor i_p,
                                     Predictor j_p,
                                     Forces &d_f,
                                    float   j_mass)
{
    double rx = j_p.r[0] - i_p.r[0];
    double ry = j_p.r[1] - i_p.r[1];
    double rz = j_p.r[2] - i_p.r[2];

    double vx = j_p.v[0] - i_p.v[0];
    double vy = j_p.v[1] - i_p.v[1];
    double vz = j_p.v[2] - i_p.v[2];

    double r2 = rx*rx + ry*ry + rz*rz + E2;
    double rinv = rsqrt(r2);
    double r2inv = rinv  * rinv;
    double r3inv = r2inv * rinv;
    double r5inv = r2inv * r3inv;
    double mr3inv = r3inv * j_mass;
    double mr5inv = r5inv * j_mass;

    double rv = rx*vx + ry*vy + rz*vz;

    d_f.a[0] += (rx * mr3inv);
    d_f.a[1] += (ry * mr3inv);
    d_f.a[2] += (rz * mr3inv);

    d_f.a1[0] += (vx * mr3inv - (3 * rv) * rx * mr5inv);
    d_f.a1[1] += (vy * mr3inv - (3 * rv) * ry * mr5inv);
    d_f.a1[2] += (vz * mr3inv - (3 * rv) * rz * mr5inv);
}

__global__ void k_update(Predictor *d_i,
                         Predictor *d_j,
                         Forces *d_fout,
                         float *d_m,
                         int n,
                         int total)
{
    int ibid = blockIdx.x;
    int jbid = blockIdx.y;
    int tid = threadIdx.x;
    int iaddr = tid + blockDim.x * ibid;
    int jstart = (n * (jbid  )) / NJBLOCK;
    int jend   = (n * (jbid+1)) / NJBLOCK;
    float mj;

    Predictor ip = d_i[iaddr];
    Forces fo;
    fo.a[0] = 0.0;
    fo.a[1] = 0.0;
    fo.a[2] = 0.0;
    fo.a1[0] = 0.0;
    fo.a1[1] = 0.0;
    fo.a1[2] = 0.0;

    for(int j=jstart; j<jend; j+=BSIZE){
        __shared__ Predictor jpshare[BSIZE];
        __syncthreads();
        Predictor *src = (Predictor *)&d_j[j];
        Predictor *dst = (Predictor *)jpshare;
        dst[      tid] = src[      tid];
        dst[BSIZE+tid] = src[BSIZE+tid];
        mj = d_m[BSIZE + tid];
        __syncthreads();

        if(jend-j < BSIZE){
            for(int jj=0; jj<jend-j; jj++){
                Predictor jp = jpshare[jj];
                k_force_calculation2(ip, jp, fo, mj);
            }
        }
        else{
            for(int jj=0; jj<BSIZE; jj++){
                Predictor jp = jpshare[jj];
                k_force_calculation2(ip, jp, fo, mj);
            }
        }
    }
    //Forces foo;
    //foo.a[0] = 1.0;
    //foo.a[1]  = 1.0;
    //foo.a[2]  = 1.0;
    //foo.a1[0] = 1.0;
    //foo.a1[1] = 1.0;
    //foo.a1[2] = 1.0;

    //if(iaddr == 0)
    //    d_fout[iaddr*NJBLOCK + jbid] = foo;
    //else
    //    d_fout[iaddr*NJBLOCK + jbid] = fo;
    d_fout[iaddr*NJBLOCK + jbid] = fo;

}
