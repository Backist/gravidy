#include "hip/hip_runtime.h"
#include "dynamics_gpu.cuh"


/*
 * @fn __host__ void gpu_init_acc_jrk()
 *
 * @brief
 *  Initial calculation of the acceleration and jerk on the GPU.
 */
__host__ void gpu_init_acc_jrk()
{
    int smem = BSIZE * 2* sizeof(double4);

    k_init_acc_jrk <<< nblocks, nthreads, smem >>> (d_r, d_v, d_f, d_m, n,e2);
    hipDeviceSynchronize();
    get_kernel_error();
}

/*
 * @fn __host__ double gpu_energy()
 *
 * @brief
 *  Energy calculation on the GPU.
 */
__host__ double gpu_energy()
{

    CUDA_SAFE_CALL(hipMemcpy(d_r,  h_r,  d4_size,hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_v,  h_v,  d4_size,hipMemcpyHostToDevice));

    gpu_timer_start();
    k_energy <<< nblocks, nthreads >>> (d_r, d_v, d_ekin, d_epot, d_m, n);
    hipDeviceSynchronize();
    float msec = gpu_timer_stop("k_energy");
    get_kernel_error();

    CUDA_SAFE_CALL(hipMemcpy(h_ekin, d_ekin, d1_size,hipMemcpyDeviceToHost));
    CUDA_SAFE_CALL(hipMemcpy(h_epot, d_epot, d1_size,hipMemcpyDeviceToHost));

    // Reduction on CPU
    ekin = 0.0;
    epot = 0.0;

    for (int i = 0; i < n; i++) {
        ekin += h_ekin[i];
        epot += h_epot[i];
    }

    return ekin + epot;
}

/*
 * @fn __host__ void gpu_update()
 *
 * @brief
 *  Gravitational interactions calculation for the nact particles
 *  using j-parallelization, and then doing a reduction.
 */
__host__ void gpu_update(int total)
{

    gtime.update_ini = omp_get_wtime();

    // Copying to the device the predicted r and v
    CUDA_SAFE_CALL(hipMemcpy(d_p, h_p, sizeof(Predictor) * n,hipMemcpyHostToDevice));

    // Fill the h_i Predictor array with the particles that we need
    // to move in this iteration
    for (int i = 0; i < total; i++) {
        int id = h_move[i];
        h_i[i] = h_p[id];
    }

    // Copy to the GPU (d_i) the preddictor host array (h_i)
    CUDA_SAFE_CALL(hipMemcpy(d_i, h_i, sizeof(Predictor) * total, hipMemcpyHostToDevice));


    // Blocks, threads and shared memory configuration
    dim3 nblocks(1 + (total-1)/BSIZE,NJBLOCK, 1);
    dim3 nthreads(BSIZE, 1, 1);
    size_t smem = BSIZE * sizeof(Predictor);

    // Kernel to update the forces for the particles in d_i
    gtime.grav_ini = omp_get_wtime();
    k_update <<< nblocks, nthreads, smem >>> (d_i, d_p, d_fout,d_m, n, total,e2);
    hipDeviceSynchronize();
    gtime.grav_end += omp_get_wtime() - gtime.grav_ini;
    get_kernel_error();

    // Blocks, threads and shared memory configuration for the reduction.
    dim3 rgrid   (total,   1, 1);
    dim3 rthreads(NJBLOCK, 1, 1);
    size_t smem2 = sizeof(Forces) * NJBLOCK + 1;

    // Kernel to reduce que temp array with the forces
    gtime.reduce_ini = omp_get_wtime();
    reduce <<< rgrid, rthreads, smem2 >>>(d_fout, d_fout_tmp, total);
    hipDeviceSynchronize();
    gtime.reduce_end += omp_get_wtime() - gtime.grav_ini;
    get_kernel_error();

    // Copy from the GPU the new forces for the d_i particles.
    CUDA_SAFE_CALL(hipMemcpy(h_fout_tmp, d_fout_tmp, sizeof(Forces) * total, hipMemcpyDeviceToHost));

    // Update forces in the host
    for (int i = 0; i < total; i++) {
        int id = h_move[i];
        h_f[id] = h_fout_tmp[i];
    }

    gtime.update_end += omp_get_wtime() - gtime.update_ini;

}
