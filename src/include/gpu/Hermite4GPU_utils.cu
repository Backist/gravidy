#include "Hermite4GPU.cuh"

void Hermite4GPU::get_kernel_error(){
    #ifdef KERNEL_ERROR_DEBUG
        std::cerr << "[Error] : ";
        std::cerr << hipGetErrorString(hipGetLastError()) << std::endl;
    #endif
}

void Hermite4GPU::gpu_timer_start(){
    hipEventRecord(start);
}

float HermiteGPU::gpu_timer_stop(string f){
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float msec = 0;
    hipEventElapsedTime(&msec, start, stop);
    #if KERNEL_TIME
    if (f != "")
        std::cout << "[Time] " << f << " : " << msec << " msec" << std::endl;
    #endif
    return msec;
}
