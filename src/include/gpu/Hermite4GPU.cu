#include "hip/hip_runtime.h"
#include "Hermite4GPU.cuh"

Hermite4GPU::~Hermite4GPU()
{
    free_arrays_device();
}

void Hermite4GPU::alloc_arrays_device()
{
    int d4_size = ns->n * sizeof(double4);
    int d1_size = ns->n * sizeof(double);
    int i1_size = ns->n * sizeof(int);
    int ff_size = ns->n * sizeof(Forces);
    int pp_size = ns->n * sizeof(Predictor);

    CUDA_SAFE_CALL(hipMalloc((void**)&ns->d_r,        d4_size));
    CUDA_SAFE_CALL(hipMalloc((void**)&ns->d_v,        d4_size));
    CUDA_SAFE_CALL(hipMalloc((void**)&ns->d_f,        ff_size));
    CUDA_SAFE_CALL(hipMalloc((void**)&ns->d_p,        pp_size));
    CUDA_SAFE_CALL(hipMalloc((void**)&ns->d_ekin,     d1_size));
    CUDA_SAFE_CALL(hipMalloc((void**)&ns->d_epot,     d1_size));
    CUDA_SAFE_CALL(hipMalloc((void**)&ns->d_t,        d1_size));
    CUDA_SAFE_CALL(hipMalloc((void**)&ns->d_dt,       d1_size));
    CUDA_SAFE_CALL(hipMalloc((void**)&ns->d_move,     i1_size));
    CUDA_SAFE_CALL(hipMalloc((void**)&ns->d_i,        pp_size));
    CUDA_SAFE_CALL(hipMalloc((void**)&ns->d_fout,     ff_size * NJBLOCK));
    CUDA_SAFE_CALL(hipMalloc((void**)&ns->d_fout_tmp, ff_size * NJBLOCK));

    CUDA_SAFE_CALL(hipMemset(ns->d_r,         0, d4_size));
    CUDA_SAFE_CALL(hipMemset(ns->d_v,         0, d4_size));
    CUDA_SAFE_CALL(hipMemset(ns->d_f,         0, ff_size));
    CUDA_SAFE_CALL(hipMemset(ns->d_p,         0, pp_size));
    CUDA_SAFE_CALL(hipMemset(ns->d_ekin,      0, d1_size));
    CUDA_SAFE_CALL(hipMemset(ns->d_epot,      0, d1_size));
    CUDA_SAFE_CALL(hipMemset(ns->d_t,         0, d1_size));
    CUDA_SAFE_CALL(hipMemset(ns->d_dt,        0, d1_size));
    CUDA_SAFE_CALL(hipMemset(ns->d_move,      0, i1_size));
    CUDA_SAFE_CALL(hipMemset(ns->d_i,         0, pp_size));
    CUDA_SAFE_CALL(hipMemset(ns->d_fout,      0, ff_size * NJBLOCK));
    CUDA_SAFE_CALL(hipMemset(ns->d_fout_tmp,  0, ff_size * NJBLOCK));

    // Extra CPU array
    ns->h_fout_tmp= new Forces[ff_size*NJBLOCK];

}

void Hermite4GPU::free_arrays_device()
{
    CUDA_SAFE_CALL(hipFree(ns->d_r));
    CUDA_SAFE_CALL(hipFree(ns->d_v));
    CUDA_SAFE_CALL(hipFree(ns->d_f));
    CUDA_SAFE_CALL(hipFree(ns->d_p));
    CUDA_SAFE_CALL(hipFree(ns->d_ekin));
    CUDA_SAFE_CALL(hipFree(ns->d_epot));
    CUDA_SAFE_CALL(hipFree(ns->d_t));
    CUDA_SAFE_CALL(hipFree(ns->d_dt));
    CUDA_SAFE_CALL(hipFree(ns->d_move));
    CUDA_SAFE_CALL(hipFree(ns->d_i));
    CUDA_SAFE_CALL(hipFree(ns->d_fout));
    CUDA_SAFE_CALL(hipFree(ns->d_fout_tmp));

    delete ns->h_fout_tmp;
}

/** Not implemented using GPU */
void Hermite4GPU::predicted_pos_vel(double ITIME)
{

    ns->gtime.prediction_ini = omp_get_wtime();
    for (int i = 0; i < ns->n; i++)
    {
        double dt  = ITIME - ns->h_t[i];
        double dt2 = (dt  * dt);
        double dt3 = (dt2 * dt);

        ns->h_p[i].r[0] = (dt3/6 * ns->h_f[i].a1[0]) + (dt2/2 * ns->h_f[i].a[0]) + (dt * ns->h_v[i].x) + ns->h_r[i].x;
        ns->h_p[i].r[1] = (dt3/6 * ns->h_f[i].a1[1]) + (dt2/2 * ns->h_f[i].a[1]) + (dt * ns->h_v[i].y) + ns->h_r[i].y;
        ns->h_p[i].r[2] = (dt3/6 * ns->h_f[i].a1[2]) + (dt2/2 * ns->h_f[i].a[2]) + (dt * ns->h_v[i].z) + ns->h_r[i].z;

        ns->h_p[i].v[0] = (dt2/2 * ns->h_f[i].a1[0]) + (dt * ns->h_f[i].a[0]) + ns->h_v[i].x;
        ns->h_p[i].v[1] = (dt2/2 * ns->h_f[i].a1[1]) + (dt * ns->h_f[i].a[1]) + ns->h_v[i].y;
        ns->h_p[i].v[2] = (dt2/2 * ns->h_f[i].a1[2]) + (dt * ns->h_f[i].a[2]) + ns->h_v[i].z;

        ns->h_p[i].m = ns->h_r[i].w;

    }
    ns->gtime.prediction_end += omp_get_wtime() - ns->gtime.prediction_ini;
}

/** Not implemented using GPU */
void Hermite4GPU::correction_pos_vel(double ITIME, int nact)
{
    ns->gtime.correction_ini = omp_get_wtime();
    for (int k = 0; k < nact; k++)
    {
        int i = ns->h_move[k];

        double dt1 = ns->h_dt[i];
        double dt2 = dt1 * dt1;
        double dt3 = dt2 * dt1;
        double dt4 = dt2 * dt2;
        double dt5 = dt4 * dt1;

        // Acceleration 2nd derivate
        ns->h_a2[i].x = (-6 * (ns->h_old[i].a[0] - ns->h_f[i].a[0] ) - dt1 * (4 * ns->h_old[i].a1[0] + 2 * ns->h_f[i].a1[0]) ) / dt2;
        ns->h_a2[i].y = (-6 * (ns->h_old[i].a[1] - ns->h_f[i].a[1] ) - dt1 * (4 * ns->h_old[i].a1[1] + 2 * ns->h_f[i].a1[1]) ) / dt2;
        ns->h_a2[i].z = (-6 * (ns->h_old[i].a[2] - ns->h_f[i].a[2] ) - dt1 * (4 * ns->h_old[i].a1[2] + 2 * ns->h_f[i].a1[2]) ) / dt2;

        // Acceleration 3rd derivate
        ns->h_a3[i].x = (12 * (ns->h_old[i].a[0] - ns->h_f[i].a[0] ) + 6 * dt1 * (ns->h_old[i].a1[0] + ns->h_f[i].a1[0]) ) / dt3;
        ns->h_a3[i].y = (12 * (ns->h_old[i].a[1] - ns->h_f[i].a[1] ) + 6 * dt1 * (ns->h_old[i].a1[1] + ns->h_f[i].a1[1]) ) / dt3;
        ns->h_a3[i].z = (12 * (ns->h_old[i].a[2] - ns->h_f[i].a[2] ) + 6 * dt1 * (ns->h_old[i].a1[2] + ns->h_f[i].a1[2]) ) / dt3;

        // Correcting position
        ns->h_r[i].x = ns->h_p[i].r[0] + (dt4/24)*ns->h_a2[i].x + (dt5/120)*ns->h_a3[i].x;
        ns->h_r[i].y = ns->h_p[i].r[1] + (dt4/24)*ns->h_a2[i].y + (dt5/120)*ns->h_a3[i].y;
        ns->h_r[i].z = ns->h_p[i].r[2] + (dt4/24)*ns->h_a2[i].z + (dt5/120)*ns->h_a3[i].z;

        // Correcting velocity
        ns->h_v[i].x = ns->h_p[i].v[0] + (dt3/6)*ns->h_a2[i].x + (dt4/24)*ns->h_a3[i].x;
        ns->h_v[i].y = ns->h_p[i].v[1] + (dt3/6)*ns->h_a2[i].y + (dt4/24)*ns->h_a3[i].y;
        ns->h_v[i].z = ns->h_p[i].v[2] + (dt3/6)*ns->h_a2[i].z + (dt4/24)*ns->h_a3[i].z;


        ns->h_t[i] = ITIME;
        double normal_dt  = nu->get_timestep_normal(i);
        normal_dt = nu->normalize_dt(normal_dt, ns->h_dt[i], ns->h_t[i], i);
        ns->h_dt[i] = normal_dt;

    }
    ns->gtime.correction_end += omp_get_wtime() - ns->gtime.correction_ini;
}

void Hermite4GPU::init_acc_jrk()
{

    CUDA_SAFE_CALL(hipMemcpy(ns->d_p,
                              ns->h_p,
                              ns->n * sizeof(Predictor),
                              hipMemcpyHostToDevice));

    k_init_acc_jrk <<< nblocks, nthreads, smem >>> (ns->d_p,
                                                    ns->d_f,
                                                    ns->n,
                                                    ns->e2);
    //get_kernel_error();

    CUDA_SAFE_CALL(hipMemcpy(ns->h_f,
                              ns->d_f,
                              ns->n * sizeof(Forces),
                              hipMemcpyDeviceToHost));
}

void Hermite4GPU::update_acc_jrk(int nact)
{
    ns->gtime.update_ini = omp_get_wtime();

    // Copying to the device the predicted r and v
    CUDA_SAFE_CALL(hipMemcpy(ns->d_p,
                              ns->h_p,
                              ns->n * sizeof(Predictor),
                              hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(ns->d_move,
                              ns->h_move,
                              ns->n * sizeof(int),
                              hipMemcpyHostToDevice));

    // Fill the h_i Predictor array with the particles that we need
    // to move in this iteration
    for (int i = 0; i < nact; i++)
    {
        int id = ns->h_move[i];
        ns->h_i[i] = ns->h_p[id];
    }

    // Copy to the GPU (d_i) the preddictor host array (h_i)
    CUDA_SAFE_CALL(hipMemcpy(ns->d_i,
                              ns->h_i,
                              nact * sizeof(Predictor),
                              hipMemcpyHostToDevice));


    // Blocks, threads and shared memory configuration
    int  nact_blocks = 1 + (nact-1)/BSIZE;
    dim3 nblocks(nact_blocks,NJBLOCK, 1);
    dim3 nthreads(BSIZE, 1, 1);

    // Kernel to update the forces for the particles in d_i
    ns->gtime.grav_ini = omp_get_wtime();
    k_update <<< nblocks, nthreads, smem >>> (ns->d_i,
                                              ns->d_p,
                                              ns->d_fout,
                                              ns->d_move,
                                              ns->n,
                                              nact,
                                              ns->e2);
    hipDeviceSynchronize();
    ns->gtime.grav_end += omp_get_wtime() - ns->gtime.grav_ini;
    //get_kernel_error();

    // Blocks, threads and shared memory configuration for the reduction.
    dim3 rgrid   (nact,   1, 1);
    dim3 rthreads(NJBLOCK, 1, 1);

    // Kernel to reduce que temp array with the forces
    ns->gtime.reduce_ini = omp_get_wtime();
    reduce <<< rgrid, rthreads, smem_reduce >>>(ns->d_fout,
                                                ns->d_fout_tmp);
    ns->gtime.reduce_end += omp_get_wtime() - ns->gtime.reduce_ini;
    //get_kernel_error();

    // Copy from the GPU the new forces for the d_i particles.
    CUDA_SAFE_CALL(hipMemcpy(ns->h_fout_tmp,
                              ns->d_fout_tmp,
                              nact * sizeof(Forces),
                              hipMemcpyDeviceToHost));

    // Update forces in the host
    for (int i = 0; i < nact; i++)
    {
        int id = ns->h_move[i];
        ns->h_f[id] = ns->h_fout_tmp[i];
    }

    ns->gtime.update_end += (omp_get_wtime() - ns->gtime.update_ini);
}

double Hermite4GPU::get_energy_gpu()
{
    CUDA_SAFE_CALL(hipMemcpy(ns->d_r, ns->h_r,  sizeof(double4) * ns->n,hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(ns->d_v, ns->h_v,  sizeof(double4) * ns->n,hipMemcpyHostToDevice));

    //gpu_timer_start();
    int nthreads = BSIZE;
    int nblocks = std::ceil(ns->n/(float)nthreads);
    k_energy <<< nblocks, nthreads >>> (ns->d_r,
                                        ns->d_v,
                                        ns->d_ekin,
                                        ns->d_epot,
                                        ns->n,
                                        ns->e2);
    hipDeviceSynchronize();
    //float msec = gpu_timer_stop("k_energy");
    //get_kernel_error();

    CUDA_SAFE_CALL(hipMemcpy(ns->h_ekin, ns->d_ekin, sizeof(double) * ns->n,hipMemcpyDeviceToHost));
    CUDA_SAFE_CALL(hipMemcpy(ns->h_epot, ns->d_epot, sizeof(double) * ns->n,hipMemcpyDeviceToHost));

    // Reduction on CPU
    ns->en.kinetic = 0.0;
    ns->en.potential = 0.0;

    for (int i = 0; i < ns->n; i++)
    {
        ns->en.kinetic   += ns->h_ekin[i];
        ns->en.potential += ns->h_epot[i];
    }
    return ns->en.kinetic + ns->en.potential;
}

/*
 * @fn k_init_acc_jrk
 *
 * @desc GPU Kernel which calculates the initial acceleration and jerk
 * of all the particles of the system.
 *
 */
__global__ void k_init_acc_jrk (Predictor *p,
                                Forces *f,
                                int n,
                                double e2)
{

    extern __shared__ Predictor sh[];

    Forces ff;
    ff.a[0]  = 0.0;
    ff.a[1]  = 0.0;
    ff.a[2]  = 0.0;
    ff.a1[0] = 0.0;
    ff.a1[1] = 0.0;
    ff.a1[2] = 0.0;

    int id = threadIdx.x + blockDim.x * blockIdx.x;
    int tx = threadIdx.x;

    if (id < n)
    {
        Predictor pred = p[id];
        int tile = 0;
        for (int i = 0; i < n; i += BSIZE)
        {
            int idx = tile * BSIZE + tx;
            sh[tx]   = p[idx];
            __syncthreads();
            for (int k = 0; k < BSIZE; k++)
            {
                k_force_calculation(pred, sh[k], ff, e2);
            }
            __syncthreads();
            tile++;
        }
        f[id] = ff;
    }
}

__device__ void k_force_calculation(Predictor i_p,
                                    Predictor j_p,
                                    Forces &f,
                                    double e2)
{
    double rx = j_p.r[0] - i_p.r[0];
    double ry = j_p.r[1] - i_p.r[1];
    double rz = j_p.r[2] - i_p.r[2];

    double vx = j_p.v[0] - i_p.v[0];
    double vy = j_p.v[1] - i_p.v[1];
    double vz = j_p.v[2] - i_p.v[2];

    double r2     = rx*rx + ry*ry + rz*rz + e2;
    double rinv   = rsqrt(r2);
    double r2inv  = rinv  * rinv;
    double r3inv  = r2inv * rinv;
    double r5inv  = r2inv * r3inv;
    double mr3inv = r3inv * j_p.m;
    double mr5inv = r5inv * j_p.m;

    double rv = rx*vx + ry*vy + rz*vz;

    f.a[0] += (rx * mr3inv);
    f.a[1] += (ry * mr3inv);
    f.a[2] += (rz * mr3inv);

    f.a1[0] += (vx * mr3inv - (3 * rv) * rx * mr5inv);
    f.a1[1] += (vy * mr3inv - (3 * rv) * ry * mr5inv);
    f.a1[2] += (vz * mr3inv - (3 * rv) * rz * mr5inv);
}

/*
 * @fn k_update()
 *
 * @brief Gravitational interaction kernel.
 */
__global__ void k_update(Predictor *i_p,
                         Predictor *j_p,
                         Forces *fout,
                         int *move,
                         int n,
                         int total,
                         double e2)
{
    int ibid = blockIdx.x;
    int jbid = blockIdx.y;
    int tid  = threadIdx.x;
    int iaddr  = tid + blockDim.x * ibid;
    int jstart = (n * (jbid  )) / NJBLOCK;
    int jend   = (n * (jbid+1)) / NJBLOCK;

    Predictor ip = i_p[iaddr];
    Forces fo;
    fo.a[0] = 0.0;
    fo.a[1] = 0.0;
    fo.a[2] = 0.0;
    fo.a1[0] = 0.0;
    fo.a1[1] = 0.0;
    fo.a1[2] = 0.0;

        for(int j=jstart; j<jend; j+=BSIZE)
        {
            __shared__ Predictor jpshare[BSIZE];
            __syncthreads();
            Predictor *src = (Predictor *)&j_p[j];
            Predictor *dst = (Predictor *)jpshare;
            dst[      tid] = src[      tid];
            dst[BSIZE+tid] = src[BSIZE+tid];
            __syncthreads();

            // If the total amount of particles is not a multiple of BSIZE
            if(jend-j < BSIZE)
            {
                #pragma unroll 4
                for(int jj=0; jj<jend-j; jj++)
                {
                    Predictor jp = jpshare[jj];
                    k_force_calculation(ip, jp, fo, e2);
                }
            }
            else
            {
                #pragma unroll 4
                for(int jj=0; jj<BSIZE; jj++)
                {
                    Predictor jp = jpshare[jj];
                    k_force_calculation(ip, jp, fo, e2);
                }
            }
        }
        fout[iaddr*NJBLOCK + jbid] = fo;
}

/*
 * @fn reduce()
 *
 * @brief Forces reduction kernel
 */
__global__ void reduce(Forces *in,
                       Forces *out)
{
    extern __shared__ Forces sdata[];

    const int xid   = threadIdx.x;
    const int bid   = blockIdx.x;
    const int iaddr = xid + blockDim.x * bid;

    sdata[xid] = in[iaddr];
    __syncthreads();

    if(xid < 8) sdata[xid] += sdata[xid + 8];
    if(xid < 4) sdata[xid] += sdata[xid + 4];
    if(xid < 2) sdata[xid] += sdata[xid + 2];
    if(xid < 1) sdata[xid] += sdata[xid + 1];

    if(xid == 0){
        out[bid] = sdata[0];
    }
}

__global__ void k_energy(double4 *r,
                         double4 *v,
                         double *ekin,
                         double *epot,
                         int n,
                         double e2)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j;
    double ekin_tmp = 0.0;

    if (i < n)
    {
        double epot_tmp = 0.0;
        for (j = i+1; j < n; j++)
        {
            double rx = r[j].x - r[i].x;
            double ry = r[j].y - r[i].y;
            double rz = r[j].z - r[i].z;
            double r2 = rx*rx + ry*ry + rz*rz + e2;

            epot_tmp -= (r[i].w * r[j].w) * rsqrt(r2);
        }

        double vx = v[i].x * v[i].x;
        double vy = v[i].y * v[i].y;
        double vz = v[i].z * v[i].z;
        double v2 = vx + vy + vz;

        ekin_tmp = 0.5 * r[i].w * v2;

        ekin[i] = ekin_tmp;
        epot[i] = epot_tmp;
    }
}

void Hermite4GPU::get_kernel_error(){
    #ifdef KERNEL_ERROR_DEBUG
        std::cerr << "[Error] : ";
        std::cerr << hipGetErrorString(hipGetLastError()) << std::endl;
    #endif
}

void Hermite4GPU::gpu_timer_start(){
    hipEventRecord(start);
}

float Hermite4GPU::gpu_timer_stop(std::string f){
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float msec = 0;
    hipEventElapsedTime(&msec, start, stop);
    #if KERNEL_TIME
    if (f != "")
        std::cout << "[Time] " << f << " : " << msec << " msec" << std::endl;
    #endif
    return msec;
}

void Hermite4GPU::force_calculation(int i, int j)
{
    double rx = ns->h_p[j].r[0] - ns->h_p[i].r[0];
    double ry = ns->h_p[j].r[1] - ns->h_p[i].r[1];
    double rz = ns->h_p[j].r[2] - ns->h_p[i].r[2];

    double vx = ns->h_p[j].v[0] - ns->h_p[i].v[0];
    double vy = ns->h_p[j].v[1] - ns->h_p[i].v[1];
    double vz = ns->h_p[j].v[2] - ns->h_p[i].v[2];

    double r2     = rx*rx + ry*ry + rz*rz + ns->e2;
    double rinv   = 1.0/sqrt(r2);
    double r2inv  = rinv  * rinv;
    double r3inv  = r2inv * rinv;
    double r5inv  = r2inv * r3inv;
    double mr3inv = r3inv * ns->h_p[j].m;
    double mr5inv = r5inv * ns->h_p[j].m;

    double rv = rx*vx + ry*vy + rz*vz;

    ns->h_f[i].a[0] += (rx * mr3inv);
    ns->h_f[i].a[1] += (ry * mr3inv);
    ns->h_f[i].a[2] += (rz * mr3inv);

    ns->h_f[i].a1[0] += (vx * mr3inv - (3 * rv ) * rx * mr5inv);
    ns->h_f[i].a1[1] += (vy * mr3inv - (3 * rv ) * ry * mr5inv);
    ns->h_f[i].a1[2] += (vz * mr3inv - (3 * rv ) * rz * mr5inv);
}

void Hermite4GPU::integration()
{
    ns->gtime.integration_ini = omp_get_wtime();

    double ATIME = 1.0e+10; // Actual integration time
    double ITIME = 0.0;     // Integration time
    int nact     = 0;       // Active particles
    int nsteps   = 0;       // Amount of steps per particles on the system
    static long long interactions = 0;


    int max_threads = omp_get_max_threads();
    omp_set_num_threads( max_threads - 1);

    init_acc_jrk();
    init_dt(ATIME);

    ns->en.ini = get_energy_gpu();   // Initial calculation of the energy of the system
    ns->en.tmp = ns->en.ini;

    //ns->hmr_time = nu->get_half_mass_relaxation_time();
    //ns->cr_time  = nu->get_crossing_time();

    logger->print_info();
    logger->print_energy_log(ITIME, ns->iterations, interactions, nsteps, ns->en.ini);

    if (ns->ops.print_all)
    {
        logger->print_all(ITIME);
    }
    if (ns->ops.print_lagrange)
    {
        nu->lagrange_radii();
        logger->print_lagrange_radii(ITIME, nu->layers_radii);
    }

    while (ITIME < ns->integration_time)
    {
        ITIME = ATIME;

        nact = find_particles_to_move(ITIME);

        save_old_acc_jrk(nact);

        predicted_pos_vel(ITIME);

        update_acc_jrk(nact);

        correction_pos_vel(ITIME, nact);

        // Update the amount of interactions counter
        interactions += nact * ns->n;

        // Find the next integration time
        next_integration_time(ATIME);


        if(std::ceil(ITIME) == ITIME)
        {
            assert(nact == ns->n);
            logger->print_energy_log(ITIME, ns->iterations, interactions, nsteps, get_energy_gpu());
            if (ns->ops.print_all)
            {
                logger->print_all(ITIME);
            }
            if (ns->ops.print_lagrange)
            {
                nu->lagrange_radii();
                logger->print_lagrange_radii(ITIME, nu->layers_radii);
            }
        }

        // Update nsteps with nact
        nsteps += nact;

        // Increase iteration counter
        ns->iterations++;
    }
    ns->gtime.integration_end =  omp_get_wtime() - ns->gtime.integration_ini;

}
