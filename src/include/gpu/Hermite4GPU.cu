#include "hip/hip_runtime.h"
#include "Hermite4GPU.cuh"

Hermite4GPU::~Hermite4GPU()
{
    free_arrays_device();
}

void Hermite4GPU::alloc_arrays_device()
{

    size_t i1_size = ns->n * sizeof(int);
    size_t d1_size = ns->n * sizeof(double);
    size_t d4_size = ns->n * sizeof(double4);
    size_t ff_size = ns->n * sizeof(Forces);
    size_t pp_size = ns->n * sizeof(Predictor);

    for(int g = 0; g < gpus; g++)
    {
        // Setting GPU
        CSC(hipSetDevice(g));

        CSC(hipMalloc((void**)&ns->d_r[g], d4_size));
        CSC(hipMalloc((void**)&ns->d_v[g], d4_size));
        CSC(hipMalloc((void**)&ns->d_f[g], ff_size));
        CSC(hipMalloc((void**)&ns->d_p[g], pp_size));
        CSC(hipMalloc((void**)&ns->d_t[g], d1_size));
        CSC(hipMalloc((void**)&ns->d_i[g], pp_size));
        CSC(hipMalloc((void**)&ns->d_dt[g], d1_size));
        CSC(hipMalloc((void**)&ns->d_ekin[g], d1_size));
        CSC(hipMalloc((void**)&ns->d_epot[g], d1_size));
        CSC(hipMalloc((void**)&ns->d_move[g], i1_size));
        CSC(hipMalloc((void**)&ns->d_fout[g], ff_size * NJBLOCK));
        CSC(hipMalloc((void**)&ns->d_fout_tmp[g], ff_size * NJBLOCK));

        CSC(hipMemset(ns->d_r[g], 0, d4_size));
        CSC(hipMemset(ns->d_v[g], 0, d4_size));
        CSC(hipMemset(ns->d_f[g], 0, ff_size));
        CSC(hipMemset(ns->d_p[g], 0, pp_size));
        CSC(hipMemset(ns->d_t[g], 0, d1_size));
        CSC(hipMemset(ns->d_i[g], 0, pp_size));
        CSC(hipMemset(ns->d_dt[g], 0, d1_size));
        CSC(hipMemset(ns->d_ekin[g], 0, d1_size));
        CSC(hipMemset(ns->d_epot[g], 0, d1_size));
        CSC(hipMemset(ns->d_move[g], 0, i1_size));
        CSC(hipMemset(ns->d_fout[g], 0, ff_size * NJBLOCK));
        CSC(hipMemset(ns->d_fout_tmp[g], 0, ff_size * NJBLOCK));

        ns->h_fout_gpu[g] = new Forces[ns->n*NJBLOCK];
    }

    // Extra CPU array
    ns->h_fout_tmp = new Forces[ns->n*NJBLOCK];

}

void Hermite4GPU::free_arrays_device()
{

    for(int g = 0; g < gpus; g++)
    {
        // Setting GPU
        CSC(hipSetDevice(g));

        CSC(hipFree(ns->d_r[g]));
        CSC(hipFree(ns->d_v[g]));
        CSC(hipFree(ns->d_f[g]));
        CSC(hipFree(ns->d_p[g]));
        CSC(hipFree(ns->d_t[g]));
        CSC(hipFree(ns->d_i[g]));
        CSC(hipFree(ns->d_dt[g]));
        CSC(hipFree(ns->d_ekin[g]));
        CSC(hipFree(ns->d_epot[g]));
        CSC(hipFree(ns->d_move[g]));
        CSC(hipFree(ns->d_fout[g]));
        CSC(hipFree(ns->d_fout_tmp[g]));
        delete ns->h_fout_gpu[g];
    }

    delete ns->h_fout_tmp;
    //delete ns->h_fout_gpu;
}

/** Not implemented using GPU */
void Hermite4GPU::predicted_pos_vel(double ITIME)
{

    ns->gtime.prediction_ini = omp_get_wtime();
    #pragma omp parallel for
    for (int i = 0; i < ns->n; i++)
    {
        double dt  = ITIME - ns->h_t[i];
        double dt2 = 0.5*(dt  * dt);
        double dt3 = 0.166666666666666*(dt * dt * dt);

        Forces ff = ns->h_f[i];
        double4 rr = ns->h_r[i];
        double4 vv = ns->h_v[i];

        ns->h_p[i].r[0] = (dt3 * ff.a1[0]) + (dt2 * ff.a[0]) + (dt * vv.x) + rr.x;
        ns->h_p[i].r[1] = (dt3 * ff.a1[1]) + (dt2 * ff.a[1]) + (dt * vv.y) + rr.y;
        ns->h_p[i].r[2] = (dt3 * ff.a1[2]) + (dt2 * ff.a[2]) + (dt * vv.z) + rr.z;

        ns->h_p[i].v[0] = (dt2 * ff.a1[0]) + (dt * ff.a[0]) + vv.x;
        ns->h_p[i].v[1] = (dt2 * ff.a1[1]) + (dt * ff.a[1]) + vv.y;
        ns->h_p[i].v[2] = (dt2 * ff.a1[2]) + (dt * ff.a[2]) + vv.z;

        ns->h_p[i].m = rr.w;

    }
    ns->gtime.prediction_end += omp_get_wtime() - ns->gtime.prediction_ini;
}

/** Not implemented using GPU */
void Hermite4GPU::correction_pos_vel(double ITIME, int nact)
{
    ns->gtime.correction_ini = omp_get_wtime();
    int i;
    #pragma omp parallel for private(i)
    for (int k = 0; k < nact; k++)
    {
        i = ns->h_move[k];

        Forces ff = ns->h_f[i];
        Forces oo = ns->h_old[i];
        Predictor pp = ns->h_p[i];

        double dt1 = ns->h_dt[i];
        double dt2 = dt1 * dt1;
        double dt3 = dt2 * dt1;
        double dt4 = dt2 * dt2;
        double dt5 = dt4 * dt1;

        double dt2inv = 1.0/dt2;
        double dt3inv = 1.0/dt3;

        double dt3_6 = 0.166666666666666*dt3;
        double dt4_24 = 0.041666666666666*dt4;
        double dt5_120 = 0.008333333333333*dt5;

        // Acceleration 2nd derivate
        ns->h_a2[i].x = (-6 * (oo.a[0] - ff.a[0] ) - dt1 * (4 * oo.a1[0] + 2 * ff.a1[0]) ) * dt2inv;
        ns->h_a2[i].y = (-6 * (oo.a[1] - ff.a[1] ) - dt1 * (4 * oo.a1[1] + 2 * ff.a1[1]) ) * dt2inv;
        ns->h_a2[i].z = (-6 * (oo.a[2] - ff.a[2] ) - dt1 * (4 * oo.a1[2] + 2 * ff.a1[2]) ) * dt2inv;

        // Acceleration 3rd derivate
        ns->h_a3[i].x = (12 * (oo.a[0] - ff.a[0] ) + 6 * dt1 * (oo.a1[0] + ff.a1[0]) ) * dt3inv;
        ns->h_a3[i].y = (12 * (oo.a[1] - ff.a[1] ) + 6 * dt1 * (oo.a1[1] + ff.a1[1]) ) * dt3inv;
        ns->h_a3[i].z = (12 * (oo.a[2] - ff.a[2] ) + 6 * dt1 * (oo.a1[2] + ff.a1[2]) ) * dt3inv;


        // Correcting position
        ns->h_r[i].x = pp.r[0] + (dt4_24)*ns->h_a2[i].x + (dt5_120)*ns->h_a3[i].x;
        ns->h_r[i].y = pp.r[1] + (dt4_24)*ns->h_a2[i].y + (dt5_120)*ns->h_a3[i].y;
        ns->h_r[i].z = pp.r[2] + (dt4_24)*ns->h_a2[i].z + (dt5_120)*ns->h_a3[i].z;

        // Correcting velocity
        ns->h_v[i].x = pp.v[0] + (dt3_6)*ns->h_a2[i].x + (dt4_24)*ns->h_a3[i].x;
        ns->h_v[i].y = pp.v[1] + (dt3_6)*ns->h_a2[i].y + (dt4_24)*ns->h_a3[i].y;
        ns->h_v[i].z = pp.v[2] + (dt3_6)*ns->h_a2[i].z + (dt4_24)*ns->h_a3[i].z;

        ns->h_t[i] = ITIME;

        double normal_dt  = nu->get_timestep_normal(i, ns->eta);
        ns->h_dt[i] = nu->normalize_dt(normal_dt, ns->h_dt[i], ns->h_t[i], i);

    }
    ns->gtime.correction_end += omp_get_wtime() - ns->gtime.correction_ini;
}

void Hermite4GPU::init_acc_jrk()
{

    size_t pp_size = ns->n * sizeof(Predictor);

    // Copying arrays to device
    for(int g = 0; g < gpus; g++)
    {
        CSC(hipSetDevice(g));

        // All this information from the predictors is needed by each device
        CSC(hipMemcpy(ns->d_p[g], ns->h_p, pp_size, hipMemcpyHostToDevice));
    }

    // Executing kernels
    for(int g = 0; g < gpus; g++)
    {
        CSC(hipSetDevice(g));

        nthreads = BSIZE;
        nblocks = std::ceil(n_part[g]/(float)nthreads);

        k_init_acc_jrk <<< nblocks, nthreads, smem >>> (ns->d_p[g],
                                                        ns->d_f[g],
                                                        ns->n,
                                                        ns->e2,
                                                        g,
                                                        n_part[g]);
        get_kernel_error();
    }

    for(int g = 0; g < gpus; g++)
    {
        CSC(hipSetDevice(g));

        size_t chunk = n_part[g]*sizeof(Forces);
        size_t slice = g*n_part[g-1];

        CSC(hipMemcpy(&ns->h_f[slice], ns->d_f[g], chunk, hipMemcpyDeviceToHost));
    }
}

void Hermite4GPU::update_acc_jrk(int nact)
{
    // Timer begin
    ns->gtime.update_ini = omp_get_wtime();

    for(int g = 0; g < gpus; g++)
    {
        if (n_part[g] > 0)
        {
            size_t pp_size = n_part[g] * sizeof(Predictor);
            int shift = g*n_part[g-1];

            CSC(hipSetDevice(g));
            // Copying to the device the predicted r and v
            CSC(hipMemcpy(ns->d_p[g], ns->h_p + shift, pp_size, hipMemcpyHostToDevice));
        }
    }

    // Fill the h_i Predictor array with the particles that we need to move
    #pragma omp parallel for
    for (int i = 0; i < nact; i++)
    {
        ns->h_i[i] = ns->h_p[ns->h_move[i]];
    }

    for(int g = 0; g < gpus; g++)
    {
        if (n_part[g] > 0)
        {
            CSC(hipSetDevice(g));
            // Copy to the GPU (d_i) the preddictor host array (h_i)
            size_t chunk = nact * sizeof(Predictor);
            CSC(hipMemcpy(ns->d_i[g], ns->h_i, chunk, hipMemcpyHostToDevice));
        }
    }

    ns->gtime.grav_ini = omp_get_wtime();
    for(int g = 0; g < gpus; g++)
    {
        if (n_part[g] > 0)
        {
            CSC(hipSetDevice(g));
            // Blocks, threads and shared memory configuration
            int  nact_blocks = 1 + (nact-1)/BSIZE;
            dim3 nblocks(nact_blocks, NJBLOCK, 1);
            dim3 nthreads(BSIZE, 1, 1);

            // Kernel to update the forces for the particles in d_i
            k_update <<< nblocks, nthreads, smem >>> (ns->d_i[g],
                                                      ns->d_p[g], // partial
                                                      ns->d_fout[g],
                                                      n_part[g], // former N
                                                      nact,
                                                      ns->e2);
        }
    }

    ns->gtime.grav_end += omp_get_wtime() - ns->gtime.grav_ini;
    get_kernel_error();

    ns->gtime.reduce_ini = omp_get_wtime();
    for(int g = 0; g < gpus; g++)
    {
        if (n_part[g] > 0)
        {
            CSC(hipSetDevice(g));
            // Blocks, threads and shared memory configuration for the reduction.
            dim3 rgrid   (nact,   1, 1);
            dim3 rthreads(NJBLOCK, 1, 1);

            // Kernel to reduce que temp array with the forces
            reduce <<< rgrid, rthreads, smem_reduce >>>(ns->d_fout[g],
                                                        ns->d_fout_tmp[g]);
        }
    }

    ns->gtime.reduce_end += omp_get_wtime() - ns->gtime.reduce_ini;
    get_kernel_error();

    for(int g = 0; g < gpus; g++)
    {
        if (n_part[g] > 0)
        {
            CSC(hipSetDevice(g));
            size_t chunk = nact*sizeof(Forces);

            // Copy from the GPU the new forces for the d_i particles.
            CSC(hipMemcpy(ns->h_fout_gpu[g], ns->d_fout_tmp[g], chunk,
                              hipMemcpyDeviceToHost));
        }
    }



    // Update forces in the host
    ns->gtime.reduce_forces_ini = omp_get_wtime();
    #pragma omp parallel for
    for (int i = 0; i < nact; i++)
    {
        int id = ns->h_move[i];
        ns->h_f[id].a[0] = 0.0;
        ns->h_f[id].a[1] = 0.0;
        ns->h_f[id].a[2] = 0.0;
        ns->h_f[id].a1[0] = 0.0;
        ns->h_f[id].a1[1] = 0.0;
        ns->h_f[id].a1[2] = 0.0;

        for(int g = 0; g < gpus; g++)
        {
            if (n_part[g] > 0)
            {
                ns->h_f[id] += ns->h_fout_gpu[g][i];
            }
        }
    }
    ns->gtime.reduce_forces_end += omp_get_wtime() - ns->gtime.reduce_forces_ini;

    // Timer end
    ns->gtime.update_end += (omp_get_wtime() - ns->gtime.update_ini);
}

double Hermite4GPU::get_energy_gpu()
{

    double time_energy_ini = omp_get_wtime();

    for(int g = 0; g < gpus; g++)
    {
        CSC(hipSetDevice(g));

        size_t d4_size = ns->n * sizeof(double4);

        CSC(hipMemcpy(ns->d_r[g], ns->h_r, d4_size, hipMemcpyHostToDevice));
        CSC(hipMemcpy(ns->d_v[g], ns->h_v, d4_size, hipMemcpyHostToDevice));
    }

    for(int g = 0; g < gpus; g++)
    {
        CSC(hipSetDevice(g));

        int nthreads = BSIZE;
        int nblocks = std::ceil(n_part[g]/(float)nthreads);
        k_energy <<< nblocks, nthreads >>> (ns->d_r[g],
                                            ns->d_v[g],
                                            ns->d_ekin[g],
                                            ns->d_epot[g],
                                            ns->n,
                                            n_part[g],
                                            g);
    }

    for(int g = 0; g < gpus; g++)
    {
        CSC(hipSetDevice(g));

        size_t chunk = n_part[g]*sizeof(double);
        size_t slice = g*n_part[g-1];

        CSC(hipMemcpy(&ns->h_ekin[slice], ns->d_ekin[g], chunk, hipMemcpyDeviceToHost));
        CSC(hipMemcpy(&ns->h_epot[slice], ns->d_epot[g], chunk, hipMemcpyDeviceToHost));
    }

    // Reduction on CPU
    ns->en.kinetic = 0.0;
    ns->en.potential = 0.0;

    for (int i = 0; i < ns->n; i++)
    {
        ns->en.kinetic   += ns->h_ekin[i];
        ns->en.potential += ns->h_epot[i];
    }
    double time_energy_end = omp_get_wtime() - time_energy_ini;
    return ns->en.kinetic + ns->en.potential;
}

/*
 * @fn k_init_acc_jr
 *
 *
 * @desc GPU Kernel which calculates the initial acceleration and jerk
 * of all the particles of the system.
 *
 */
__global__ void k_init_acc_jrk (Predictor *p,
                                Forces *f,
                                int n,
                                double e2,
                                int dev,
                                int dev_size)
{

    extern __shared__ Predictor sh[];

    Forces ff;
    ff.a[0]  = 0.0;
    ff.a[1]  = 0.0;
    ff.a[2]  = 0.0;
    ff.a1[0] = 0.0;
    ff.a1[1] = 0.0;
    ff.a1[2] = 0.0;

    int id = threadIdx.x + blockDim.x * blockIdx.x;
    int tx = threadIdx.x;

    if (id < dev_size)
    {
      Predictor pred = p[id+(dev*dev_size)];
      //Predictor pred = p[id];
      int tile = 0;
      for (int i = 0; i < n; i += BSIZE)
      {
          int idx = tile * BSIZE + tx;
          sh[tx]   = p[idx];
          __syncthreads();
          for (int k = 0; k < BSIZE; k++)
          {
              k_force_calculation(pred, sh[k], ff, e2);
          }
          __syncthreads();
          tile++;
      }
      f[id] = ff;
    }
}

__device__ void k_force_calculation(Predictor i_p,
                                    Predictor j_p,
                                    Forces &f,
                                    double e2)
{
    double rx = j_p.r[0] - i_p.r[0];
    double ry = j_p.r[1] - i_p.r[1];
    double rz = j_p.r[2] - i_p.r[2];

    double vx = j_p.v[0] - i_p.v[0];
    double vy = j_p.v[1] - i_p.v[1];
    double vz = j_p.v[2] - i_p.v[2];

    double r2     = rx*rx + ry*ry + rz*rz + e2;
    double rinv   = rsqrt(r2);
    double r2inv  = rinv  * rinv;
    double r3inv  = r2inv * rinv;
    double r5inv  = r2inv * r3inv;
    double mr3inv = r3inv * j_p.m;
    double mr5inv = r5inv * j_p.m;

    double rv = rx*vx + ry*vy + rz*vz;

    f.a[0] += (rx * mr3inv);
    f.a[1] += (ry * mr3inv);
    f.a[2] += (rz * mr3inv);

    f.a1[0] += (vx * mr3inv - (3 * rv) * rx * mr5inv);
    f.a1[1] += (vy * mr3inv - (3 * rv) * ry * mr5inv);
    f.a1[2] += (vz * mr3inv - (3 * rv) * rz * mr5inv);
}

/*
 * @fn k_update()
 *
 * @brief Gravitational interaction kernel.
 */
__global__ void k_update(Predictor *i_p,
                         Predictor *j_p,
                         Forces *fout,
                         int n,
                         int total,
                         double e2)
{
    int ibid = blockIdx.x;
    int jbid = blockIdx.y;
    int tid  = threadIdx.x;
    int iaddr  = tid + blockDim.x * ibid;
    int jstart = (n * (jbid  )) / NJBLOCK;
    int jend   = (n * (jbid+1)) / NJBLOCK;

    Predictor ip = i_p[iaddr];
    Forces fo;
    fo.a[0] = 0.0;
    fo.a[1] = 0.0;
    fo.a[2] = 0.0;
    fo.a1[0] = 0.0;
    fo.a1[1] = 0.0;
    fo.a1[2] = 0.0;

        for(int j=jstart; j<jend; j+=BSIZE)
        {
            __shared__ Predictor jpshare[BSIZE];
            __syncthreads();
            Predictor *src = (Predictor *)&j_p[j];
            Predictor *dst = (Predictor *)jpshare;
            dst[      tid] = src[      tid];
            dst[BSIZE+tid] = src[BSIZE+tid];
            __syncthreads();

            // If the total amount of particles is not a multiple of BSIZE
            if(jend-j < BSIZE)
            {
                #pragma unroll 4
                for(int jj=0; jj<jend-j; jj++)
                {
                    Predictor jp = jpshare[jj];
                    k_force_calculation(ip, jp, fo, e2);
                }
            }
            else
            {
                #pragma unroll 4
                for(int jj=0; jj<BSIZE; jj++)
                {
                    Predictor jp = jpshare[jj];
                    k_force_calculation(ip, jp, fo, e2);
                }
            }
        }
        fout[iaddr*NJBLOCK + jbid] = fo;
}

/*
 * @fn reduce()
 *
 * @brief Forces reduction kernel
 */
__global__ void reduce(Forces *in,
                       Forces *out)
{
    extern __shared__ Forces sdata[];

    const int xid   = threadIdx.x;
    const int bid   = blockIdx.x;
    const int iaddr = xid + blockDim.x * bid;

    sdata[xid] = in[iaddr];
    __syncthreads();

    if(xid < 8) sdata[xid] += sdata[xid + 8];
    if(xid < 4) sdata[xid] += sdata[xid + 4];
    if(xid < 2) sdata[xid] += sdata[xid + 2];
    if(xid < 1) sdata[xid] += sdata[xid + 1];

    if(xid == 0){
        out[bid] = sdata[0];
    }
}

__global__ void k_energy(double4 *r,
                         double4 *v,
                         double *ekin,
                         double *epot,
                         int n,
                         int dev_size,
                         int dev)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j;
    double ekin_tmp = 0.0;
    int id = i+dev*dev_size;

    if (i < dev_size)
    {
        double epot_tmp = 0.0;
        double4 ri = r[id];
        double4 vi = v[id];
        for (j = id+1; j < n; j++)
        {
            double rx = r[j].x - ri.x;
            double ry = r[j].y - ri.y;
            double rz = r[j].z - ri.z;
            double r2 = rx*rx + ry*ry + rz*rz;

            epot_tmp -= (ri.w * r[j].w) * rsqrt(r2);
        }

        double vx = vi.x * vi.x;
        double vy = vi.y * vi.y;
        double vz = vi.z * vi.z;
        double v2 = vx + vy + vz;

        ekin_tmp = 0.5 * ri.w * v2;

        ekin[i] = ekin_tmp;
        epot[i] = epot_tmp;
    }
}

void Hermite4GPU::get_kernel_error(){
    #ifdef KERNEL_ERROR_DEBUG
        std::cerr << "[Error] : ";
        std::cerr << hipGetErrorString(hipGetLastError()) << std::endl;
    #endif
}

void Hermite4GPU::gpu_timer_start(){
    hipEventRecord(start);
}

float Hermite4GPU::gpu_timer_stop(std::string f){
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float msec = 0;
    hipEventElapsedTime(&msec, start, stop);
    #if KERNEL_TIME
    if (f != "")
        std::cout << "[Time] " << f << " : " << msec << " msec" << std::endl;
    #endif
    return msec;
}

void Hermite4GPU::force_calculation(Predictor pi, Predictor pj, Forces &fi)
{
    double rx = pj.r[0] - pi.r[0];
    double ry = pj.r[1] - pi.r[1];
    double rz = pj.r[2] - pi.r[2];

    double vx = pj.v[0] - pi.v[0];
    double vy = pj.v[1] - pi.v[1];
    double vz = pj.v[2] - pi.v[2];

    double r2     = rx*rx + ry*ry + rz*rz + ns->e2;
    double rinv   = 1.0/sqrt(r2);
    double r2inv  = rinv  * rinv;
    double r3inv  = r2inv * rinv;
    double r5inv  = r2inv * r3inv;
    double mr3inv = r3inv * pj.m;
    double mr5inv = r5inv * pj.m;

    double rv = rx*vx + ry*vy + rz*vz;

    fi.a[0] += (rx * mr3inv);
    fi.a[1] += (ry * mr3inv);
    fi.a[2] += (rz * mr3inv);

    fi.a1[0] += (vx * mr3inv - (3 * rv ) * rx * mr5inv);
    fi.a1[1] += (vy * mr3inv - (3 * rv ) * ry * mr5inv);
    fi.a1[2] += (vz * mr3inv - (3 * rv ) * rz * mr5inv);
}

void Hermite4GPU::integration()
{
    ns->gtime.integration_ini = omp_get_wtime();

    double ATIME = 1.0e+10; // Actual integration time
    double ITIME = ns->snapshot_time;     // Integration time
    int nact     = 0;       // Active particles
    int nsteps   = 0;       // Amount of steps per particles on the system
    static long long interactions = 0;


    int max_threads = omp_get_max_threads();
    omp_set_num_threads( max_threads - 1);

    init_acc_jrk();
    init_dt(ATIME, ETA_S, ITIME);

    ns->en.ini = get_energy_gpu();   // Initial calculation of the energy of the system
    ns->en.tmp = ns->en.ini;

    //ns->hmr_time = nu->get_half_mass_relaxation_time();
    //ns->cr_time  = nu->get_crossing_time();

    logger->print_info();
    logger->write_info();
    logger->print_energy_log(ITIME, ns->iterations, interactions, nsteps, ns->en.ini);

    int snap_number = ns->snapshot_number;
    logger->write_snapshot(snap_number, ITIME);
    snap_number++;

    if (ns->ops.print_all)
    {
        logger->print_all(ITIME);
    }
    if (ns->ops.print_lagrange)
    {
        nu->lagrange_radii();
        logger->print_lagrange_radii(ITIME, nu->layers_radii);
    }

    while (ITIME < ns->integration_time)
    {
        ITIME = ATIME;

        nact = find_particles_to_move(ITIME);

        save_old_acc_jrk(nact);

        predicted_pos_vel(ITIME);

        update_acc_jrk(nact);

        correction_pos_vel(ITIME, nact);

        // Update the amount of interactions counter
        interactions += nact * ns->n;

        // Find the next integration time
        next_integration_time(ATIME);


        if(nact == ns->n)
        {
            //assert(nact == ns->n);
            logger->print_energy_log(ITIME, ns->iterations, interactions, nsteps, get_energy_gpu());
            if (ns->ops.print_all)
            {
                logger->print_all(ITIME);
            }
            if (ns->ops.print_lagrange)
            {
                nu->lagrange_radii();
                logger->print_lagrange_radii(ITIME, nu->layers_radii);
            }
            logger->write_snapshot(snap_number, ITIME);
            snap_number++;
        }

        // Update nsteps with nact
        nsteps += nact;

        // Increase iteration counter
        ns->iterations++;

    }
    ns->gtime.integration_end =  omp_get_wtime() - ns->gtime.integration_ini;
    logger->write_snapshot(snap_number, ITIME);
    //logger->add_info(std::string("SnapshotNumber:"), std::to_string(snap_number));
    logger->add_info(std::string("SnapshotNumber:"), std::string(SSTR(snap_number)));
}
