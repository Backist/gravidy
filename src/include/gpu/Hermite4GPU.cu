#include "hip/hip_runtime.h"
#include "Hermite4GPU.cuh"

Hermite4GPU::~Hermite4GPU()
{
    free_arrays_device();
}

void Hermite4GPU::alloc_arrays_device()
{

    int d4_size = ns->n * sizeof(double4);
    int d1_size = ns->n * sizeof(double);
    int i1_size = ns->n * sizeof(int);
    int ff_size = ns->n * sizeof(Forces);
    int pp_size = ns->n * sizeof(Predictor);

    for(int g = 0; g < gpus; g++)
    {
        // Setting GPU
        CSC(hipSetDevice(g));

        CSC(hipMalloc((void**)&ns->d_r[g],        d4_size));
        CSC(hipMalloc((void**)&ns->d_v[g],        d4_size));
        CSC(hipMalloc((void**)&ns->d_f[g],        ff_size));
        CSC(hipMalloc((void**)&ns->d_p[g],        pp_size));
        CSC(hipMalloc((void**)&ns->d_ekin[g],     d1_size));
        CSC(hipMalloc((void**)&ns->d_epot[g],     d1_size));
        CSC(hipMalloc((void**)&ns->d_t[g],        d1_size));
        CSC(hipMalloc((void**)&ns->d_dt[g],       d1_size));
        CSC(hipMalloc((void**)&ns->d_move[g],     i1_size));
        CSC(hipMalloc((void**)&ns->d_i[g],        pp_size));
        CSC(hipMalloc((void**)&ns->d_fout[g],     ff_size * NJBLOCK));
        CSC(hipMalloc((void**)&ns->d_fout_tmp[g], ff_size * NJBLOCK));

        CSC(hipMemset(ns->d_r[g],         0, d4_size));
        CSC(hipMemset(ns->d_v[g],         0, d4_size));
        CSC(hipMemset(ns->d_f[g],         0, ff_size));
        CSC(hipMemset(ns->d_p[g],         0, pp_size));
        CSC(hipMemset(ns->d_ekin[g],      0, d1_size));
        CSC(hipMemset(ns->d_epot[g],      0, d1_size));
        CSC(hipMemset(ns->d_t[g],         0, d1_size));
        CSC(hipMemset(ns->d_dt[g],        0, d1_size));
        CSC(hipMemset(ns->d_move[g],      0, i1_size));
        CSC(hipMemset(ns->d_i[g],         0, pp_size));
        CSC(hipMemset(ns->d_eout[g],      0, ff_size * NJBLOCK));
        CSC(hipMemset(ns->d_fout_tmp[g],  0, ff_size * NJBLOCK));
    }

    // Extra CPU array
    ns->h_fout_tmp= new Forces[ff_size*NJBLOCK];

}

void Hermite4GPU::free_arrays_device()
{

    for(int g = 0; g < gpus; g++)
    {
        // Setting GPU
        CSC(hipSetDevice(g));

        CSC(hipFree(ns->d_r[g]));
        CSC(hipFree(ns->d_v[g]));
        CSC(hipFree(ns->d_f[g]));
        CSC(hipFree(ns->d_p[g]));
        CSC(hipFree(ns->d_ekin[g]));
        CSC(hipFree(ns->d_epot[g]));
        CSC(hipFree(ns->d_t[g]));
        CSC(hipFree(ns->d_dt[g]));
        CSC(hipFree(ns->d_move[g]));
        CSC(hipFree(ns->d_i[g]));
        CSC(hipFree(ns->d_fout[g]));
        CSC(hipFree(ns->d_fout_tmp[g]));
    }

    delete ns->h_fout_tmp;
}

/** Not implemented using GPU */
void Hermite4GPU::predicted_pos_vel(double ITIME)
{

    ns->gtime.prediction_ini = omp_get_wtime();
    #pragma omp parallel for
    for (int i = 0; i < ns->n; i++)
    {
        double dt  = ITIME - ns->h_t[i];
        double dt2 = (dt  * dt);
        double dt3 = (dt2 * dt);

        ns->h_p[i].r[0] = (dt3/6 * ns->h_f[i].a1[0]) + (dt2/2 * ns->h_f[i].a[0]) + (dt * ns->h_v[i].x) + ns->h_r[i].x;
        ns->h_p[i].r[1] = (dt3/6 * ns->h_f[i].a1[1]) + (dt2/2 * ns->h_f[i].a[1]) + (dt * ns->h_v[i].y) + ns->h_r[i].y;
        ns->h_p[i].r[2] = (dt3/6 * ns->h_f[i].a1[2]) + (dt2/2 * ns->h_f[i].a[2]) + (dt * ns->h_v[i].z) + ns->h_r[i].z;

        ns->h_p[i].v[0] = (dt2/2 * ns->h_f[i].a1[0]) + (dt * ns->h_f[i].a[0]) + ns->h_v[i].x;
        ns->h_p[i].v[1] = (dt2/2 * ns->h_f[i].a1[1]) + (dt * ns->h_f[i].a[1]) + ns->h_v[i].y;
        ns->h_p[i].v[2] = (dt2/2 * ns->h_f[i].a1[2]) + (dt * ns->h_f[i].a[2]) + ns->h_v[i].z;

        ns->h_p[i].m = ns->h_r[i].w;

    }
    ns->gtime.prediction_end += omp_get_wtime() - ns->gtime.prediction_ini;
}

/** Not implemented using GPU */
void Hermite4GPU::correction_pos_vel(double ITIME, int nact)
{
    ns->gtime.correction_ini = omp_get_wtime();
    int i;
    #pragma omp parallel for private(i)
    for (int k = 0; k < nact; k++)
    {
        i = ns->h_move[k];

        double dt1 = ns->h_dt[i];
        double dt2 = dt1 * dt1;
        double dt3 = dt2 * dt1;
        double dt4 = dt2 * dt2;
        double dt5 = dt4 * dt1;

        // Acceleration 2nd derivate
        ns->h_a2[i].x = (-6 * (ns->h_old[i].a[0] - ns->h_f[i].a[0] ) - dt1 * (4 * ns->h_old[i].a1[0] + 2 * ns->h_f[i].a1[0]) ) / dt2;
        ns->h_a2[i].y = (-6 * (ns->h_old[i].a[1] - ns->h_f[i].a[1] ) - dt1 * (4 * ns->h_old[i].a1[1] + 2 * ns->h_f[i].a1[1]) ) / dt2;
        ns->h_a2[i].z = (-6 * (ns->h_old[i].a[2] - ns->h_f[i].a[2] ) - dt1 * (4 * ns->h_old[i].a1[2] + 2 * ns->h_f[i].a1[2]) ) / dt2;

        // Acceleration 3rd derivate
        ns->h_a3[i].x = (12 * (ns->h_old[i].a[0] - ns->h_f[i].a[0] ) + 6 * dt1 * (ns->h_old[i].a1[0] + ns->h_f[i].a1[0]) ) / dt3;
        ns->h_a3[i].y = (12 * (ns->h_old[i].a[1] - ns->h_f[i].a[1] ) + 6 * dt1 * (ns->h_old[i].a1[1] + ns->h_f[i].a1[1]) ) / dt3;
        ns->h_a3[i].z = (12 * (ns->h_old[i].a[2] - ns->h_f[i].a[2] ) + 6 * dt1 * (ns->h_old[i].a1[2] + ns->h_f[i].a1[2]) ) / dt3;

        // Correcting position
        ns->h_r[i].x = ns->h_p[i].r[0] + (dt4/24)*ns->h_a2[i].x + (dt5/120)*ns->h_a3[i].x;
        ns->h_r[i].y = ns->h_p[i].r[1] + (dt4/24)*ns->h_a2[i].y + (dt5/120)*ns->h_a3[i].y;
        ns->h_r[i].z = ns->h_p[i].r[2] + (dt4/24)*ns->h_a2[i].z + (dt5/120)*ns->h_a3[i].z;

        // Correcting velocity
        ns->h_v[i].x = ns->h_p[i].v[0] + (dt3/6)*ns->h_a2[i].x + (dt4/24)*ns->h_a3[i].x;
        ns->h_v[i].y = ns->h_p[i].v[1] + (dt3/6)*ns->h_a2[i].y + (dt4/24)*ns->h_a3[i].y;
        ns->h_v[i].z = ns->h_p[i].v[2] + (dt3/6)*ns->h_a2[i].z + (dt4/24)*ns->h_a3[i].z;

        ns->h_t[i] = ITIME;
        double normal_dt  = nu->get_timestep_normal(i, ns->eta);
        normal_dt = nu->normalize_dt(normal_dt, ns->h_dt[i], ns->h_t[i], i);
        ns->h_dt[i] = normal_dt;

    }
    ns->gtime.correction_end += omp_get_wtime() - ns->gtime.correction_ini;
}

void Hermite4GPU::init_acc_jrk()
{


    // Copying arrays to device
    for(int g = 0; g < gpus; g++)
    {

        CSC(hipSetDevice(g));
        std::cout << "Copying predictor in GPU: " << g << std::endl;

        // All this information from the predictors is needed by each device
        CSC(hipMemcpy(ns->d_p[g],
                                  ns->h_p,
                                  ns->n * sizeof(Predictor),
                                  hipMemcpyHostToDevice));
    }

    // Executing kernels
    for(int g = 0; g < gpus; g++)
    {
        CSC(hipSetDevice(g));

        nthreads = BSIZE;
        nblocks = std::ceil(n_part[g]/(float)nthreads);
        //std::cout << "Kernel in GPU: " << g << " | ";
        //std::cout << "Nthreads: " << nthreads <<  " | ";
        //std::cout << "Nblocks: " << nblocks << std::endl;

        k_init_acc_jrk <<< nblocks, nthreads, smem >>> (ns->d_p[g],
                                                        ns->d_f[g],
                                                        ns->n,
                                                        ns->e2,
                                                        g,
                                                        n_part[g]);
        get_kernel_error();
    }


    for(int g = 0; g < gpus; g++)
    {
        CSC(hipSetDevice(g));
        hipDeviceSynchronize();
        std::cerr << hipGetErrorString(hipGetLastError()) << std::endl;
    }

    for(int g = 0; g < gpus; g++)
    {
        CSC(hipSetDevice(g));

        size_t chunk = n_part[g]*sizeof(Forces);
        //size_t chunk = ns->n*sizeof(Forces);
        size_t slice = g*n_part[g];

        CSC(hipMemcpy(&ns->h_f[slice],
                                  ns->d_f[g],
                                  chunk,
                                  hipMemcpyDeviceToHost));
    }
    // TMP
    CSC(hipSetDevice(0));
}

void Hermite4GPU::update_acc_jrk(int nact)
{
    std::cout << "Nact: " << nact << std::endl;

    //std::cout << "Update_acc_jrk: " << nact << std::endl;
    // Timer begin
    ns->gtime.update_ini = omp_get_wtime();


    for(int g = 0; g < gpus; g++)
    {
        CSC(hipSetDevice(g));
        // Copying to the device the predicted r and v
        CSC(hipMemcpy(ns->d_p[g], ns->h_p, ns->n * sizeof(Predictor), hipMemcpyHostToDevice));
        CSC(hipMemcpy(ns->d_move[g], ns->h_move, ns->n * sizeof(int), hipMemcpyHostToDevice));
    }

    // Fill the h_i Predictor array with the particles that we need to move
    //#pragma omp parallel for
    for (int i = 0; i < nact; i++)
    {
        std::cout << ns->h_move[i] << "( " << ns->h_p[ns->h_move[i]].r[0]  << ") ";

        ns->h_i[i] = ns->h_p[ns->h_move[i]];
    }
    std::cout << std::endl;

    std::cout << "Old forces " << std::endl;
    for (int i = 0; i < nact; i++)
    {
        std::cout << ns->h_move[i] << " | " << ns->h_f[ns->h_move[i]].a[0] << std::endl;
    }

    /*************************************************************************/
    // Split nact into the amount of GPUs
    int g_nact[gpus];

    if (nact % gpus == 0)
    {
        int size = nact/gpus;
        for ( int g = 0; g < gpus; g++)
            g_nact[g] = size;
    }
    else
    {
        int size = std::ceil(nact/(float)gpus);
        for ( int g = 0; g < gpus; g++)
        {
            if (nact - size*(g+1) > 0)
                g_nact[g] = size;
            else
                g_nact[g] = nact - size*g;
        }
    }
    /*************************************************************************/

    for(int g = 0; g < gpus; g++)
    {
        CSC(hipSetDevice(g));

        if (g_nact[g] > 0)
        {
            // Copy to the GPU (d_i) the preddictor host array (h_i)
            size_t chunk = g_nact[g] * sizeof(Predictor);
            int shift = g*g_nact[g];
            printf("Copying the particles: ");
            for (int oo=0 ; oo < g_nact[g]; oo++)
            {
                std::cout << ns->h_i[oo].r[0] << " ";
            }
            std::cout << std::endl;
            CSC(hipMemcpy(ns->d_i[g], ns->h_i + shift, chunk, hipMemcpyHostToDevice));
        }
        else
        {
            //std::cout << "GPU " << g << " is not being used, due to a lack of nact" << std::endl;
        }
    }

    ns->gtime.grav_ini = omp_get_wtime();
    for(int g = 0; g < gpus; g++)
    {
        CSC(hipSetDevice(g));
        if (g_nact[g] > 0)
        {
            // Blocks, threads and shared memory configuration
            int  nact_blocks = 1 + (g_nact[g]-1)/BSIZE;
            dim3 nblocks(nact_blocks, NJBLOCK, 1);
            dim3 nthreads(BSIZE, 1, 1);

            printf("GPU %d = nact_blocks: %d, nblocks: (%d, %d, %d), nthreads (%d, %d, %d)\n",
                g, nact_blocks, nblocks.x, nblocks.y, nblocks.z, nthreads.x, nthreads.y, nthreads.z);
            // Kernel to update the forces for the particles in d_i
            k_update <<< nblocks, nthreads, smem >>> (ns->d_i[g],
                                                      ns->d_p[g],
                                                      ns->d_fout[g],
                                                      ns->d_move[g],
                                                      ns->n,
                                                      nact,
                                                      ns->e2);
            hipDeviceSynchronize();
            //std::cerr << hipGetErrorString(hipGetLastError()) << std::endl;
        }
    }


    ns->gtime.grav_end += omp_get_wtime() - ns->gtime.grav_ini;
    get_kernel_error();

    ns->gtime.reduce_ini = omp_get_wtime();

    for(int g = 0; g < gpus; g++)
    {
        CSC(hipSetDevice(g));
        if (g_nact[g] > 0)
        {
            // Blocks, threads and shared memory configuration for the reduction.
            dim3 rgrid   (g_nact[g],   1, 1);
            dim3 rthreads(NJBLOCK, 1, 1);

            // Kernel to reduce que temp array with the forces
            reduce <<< rgrid, rthreads, smem_reduce >>>(ns->d_fout[g],
                                                        ns->d_fout_tmp[g]);
            //std::cerr << hipGetErrorString(hipGetLastError()) << std::endl;
        }
    }

    ns->gtime.reduce_end += omp_get_wtime() - ns->gtime.reduce_ini;
    get_kernel_error();

    for(int g = 0; g < gpus; g++)
    {
        CSC(hipSetDevice(g));
        if (g_nact[g] > 0)
        {

            //size_t chunk = g_nact[g]*sizeof(Forces);
            size_t chunk = nact*sizeof(Forces);
            size_t slice = g*g_nact[g];

            // Copy from the GPU the new forces for the d_i particles.
            //CSC(hipMemcpy(&ns->h_fout_tmp[slice], ns->d_fout_tmp[g], chunk,
            //                  hipMemcpyDeviceToHost));
            CSC(hipMemcpy(ns->h_fout_tmp, ns->d_fout_tmp[g], chunk,
                              hipMemcpyDeviceToHost));
            //std::cerr << hipGetErrorString(hipGetLastError()) << std::endl;
        }
    }

    // Update forces in the host
    //#pragma omp parallel for
    for (int i = 0; i < nact; i++)
    {
        ns->h_f[ns->h_move[i]] = ns->h_fout_tmp[ns->h_move[i]];
    }

    std::cout << "New forces " << std::endl;
    for (int i = 0; i < nact; i++)
    {
        std::cout << ns->h_move[i] << " | " << ns->h_f[ns->h_move[i]].a[0] << std::endl;
    }

    // Timer end
    ns->gtime.update_end += (omp_get_wtime() - ns->gtime.update_ini);
    CSC(hipSetDevice(0));
    getchar();
}

double Hermite4GPU::get_energy_gpu()
{
    CSC(hipMemcpy(ns->d_r[0], ns->h_r,  sizeof(double4) * ns->n,hipMemcpyHostToDevice));
    CSC(hipMemcpy(ns->d_v[0], ns->h_v,  sizeof(double4) * ns->n,hipMemcpyHostToDevice));

    //gpu_timer_start();
    int nthreads = BSIZE;
    int nblocks = std::ceil(ns->n/(float)nthreads);
    k_energy <<< nblocks, nthreads >>> (ns->d_r[0],
                                        ns->d_v[0],
                                        ns->d_ekin[0],
                                        ns->d_epot[0],
                                        ns->n,
                                        ns->e2);
    hipDeviceSynchronize();
    //float msec = gpu_timer_stop("k_energy");
    get_kernel_error();

    CSC(hipMemcpy(ns->h_ekin, ns->d_ekin[0], sizeof(double) * ns->n,hipMemcpyDeviceToHost));
    CSC(hipMemcpy(ns->h_epot, ns->d_epot[0], sizeof(double) * ns->n,hipMemcpyDeviceToHost));

    // Reduction on CPU
    ns->en.kinetic = 0.0;
    ns->en.potential = 0.0;

    for (int i = 0; i < ns->n; i++)
    {
        ns->en.kinetic   += ns->h_ekin[i];
        ns->en.potential += ns->h_epot[i];
    }
    return ns->en.kinetic + ns->en.potential;
}

/*
 * @fn k_init_acc_jrk
 *
 * @desc GPU Kernel which calculates the initial acceleration and jerk
 * of all the particles of the system.
 *
 */
__global__ void k_init_acc_jrk (Predictor *p,
                                Forces *f,
                                int n,
                                double e2,
                                int dev,
                                int dev_size)
{

    extern __shared__ Predictor sh[];

    Forces ff;
    ff.a[0]  = 0.0;
    ff.a[1]  = 0.0;
    ff.a[2]  = 0.0;
    ff.a1[0] = 0.0;
    ff.a1[1] = 0.0;
    ff.a1[2] = 0.0;

    int id = threadIdx.x + blockDim.x * blockIdx.x;
    int tx = threadIdx.x;

    //if (id < n)
    if (id < dev_size)
    {
      Predictor pred = p[id+(dev*dev_size)];
      //Predictor pred = p[id];
      int tile = 0;
      for (int i = 0; i < n; i += BSIZE)
      {
          int idx = tile * BSIZE + tx;
          sh[tx]   = p[idx];
          __syncthreads();
          for (int k = 0; k < BSIZE; k++)
          {
              k_force_calculation(pred, sh[k], ff, e2);
          }
          __syncthreads();
          tile++;
      }
      //f[id+(dev*dev_size)] = ff;
      f[id] = ff;
    }
}

__device__ void k_force_calculation(Predictor i_p,
                                    Predictor j_p,
                                    Forces &f,
                                    double e2)
{
    double rx = j_p.r[0] - i_p.r[0];
    double ry = j_p.r[1] - i_p.r[1];
    double rz = j_p.r[2] - i_p.r[2];

    double vx = j_p.v[0] - i_p.v[0];
    double vy = j_p.v[1] - i_p.v[1];
    double vz = j_p.v[2] - i_p.v[2];

    double r2     = rx*rx + ry*ry + rz*rz + e2;
    double rinv   = rsqrt(r2);
    double r2inv  = rinv  * rinv;
    double r3inv  = r2inv * rinv;
    double r5inv  = r2inv * r3inv;
    double mr3inv = r3inv * j_p.m;
    double mr5inv = r5inv * j_p.m;

    double rv = rx*vx + ry*vy + rz*vz;

    f.a[0] += (rx * mr3inv);
    f.a[1] += (ry * mr3inv);
    f.a[2] += (rz * mr3inv);

    f.a1[0] += (vx * mr3inv - (3 * rv) * rx * mr5inv);
    f.a1[1] += (vy * mr3inv - (3 * rv) * ry * mr5inv);
    f.a1[2] += (vz * mr3inv - (3 * rv) * rz * mr5inv);
}

/*
 * @fn k_update()
 *
 * @brief Gravitational interaction kernel.
 */
__global__ void k_update(Predictor *i_p,
                         Predictor *j_p,
                         Forces *fout,
                         int *move,
                         int n,
                         int total,
                         double e2)
{
    int ibid = blockIdx.x;
    int jbid = blockIdx.y;
    int tid  = threadIdx.x;
    int iaddr  = tid + blockDim.x * ibid;
    int jstart = (n * (jbid  )) / NJBLOCK;
    int jend   = (n * (jbid+1)) / NJBLOCK;

    Predictor ip = i_p[iaddr];
    Forces fo;
    fo.a[0] = 0.0;
    fo.a[1] = 0.0;
    fo.a[2] = 0.0;
    fo.a1[0] = 0.0;
    fo.a1[1] = 0.0;
    fo.a1[2] = 0.0;

        for(int j=jstart; j<jend; j+=BSIZE)
        {
            __shared__ Predictor jpshare[BSIZE];
            __syncthreads();
            Predictor *src = (Predictor *)&j_p[j];
            Predictor *dst = (Predictor *)jpshare;
            dst[      tid] = src[      tid];
            dst[BSIZE+tid] = src[BSIZE+tid];
            __syncthreads();

            // If the total amount of particles is not a multiple of BSIZE
            if(jend-j < BSIZE)
            {
                #pragma unroll 4
                for(int jj=0; jj<jend-j; jj++)
                {
                    Predictor jp = jpshare[jj];
                    k_force_calculation(ip, jp, fo, e2);
                }
            }
            else
            {
                #pragma unroll 4
                for(int jj=0; jj<BSIZE; jj++)
                {
                    Predictor jp = jpshare[jj];
                    k_force_calculation(ip, jp, fo, e2);
                }
            }
        }
        fout[iaddr*NJBLOCK + jbid] = fo;
}

/*
 * @fn reduce()
 *
 * @brief Forces reduction kernel
 */
__global__ void reduce(Forces *in,
                       Forces *out)
{
    extern __shared__ Forces sdata[];

    const int xid   = threadIdx.x;
    const int bid   = blockIdx.x;
    const int iaddr = xid + blockDim.x * bid;

    sdata[xid] = in[iaddr];
    __syncthreads();

    if(xid < 8) sdata[xid] += sdata[xid + 8];
    if(xid < 4) sdata[xid] += sdata[xid + 4];
    if(xid < 2) sdata[xid] += sdata[xid + 2];
    if(xid < 1) sdata[xid] += sdata[xid + 1];

    if(xid == 0){
        out[bid] = sdata[0];
    }
}

__global__ void k_energy(double4 *r,
                         double4 *v,
                         double *ekin,
                         double *epot,
                         int n,
                         double e2)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j;
    double ekin_tmp = 0.0;

    if (i < n)
    {
        double epot_tmp = 0.0;
        for (j = i+1; j < n; j++)
        {
            double rx = r[j].x - r[i].x;
            double ry = r[j].y - r[i].y;
            double rz = r[j].z - r[i].z;
            double r2 = rx*rx + ry*ry + rz*rz + e2;

            epot_tmp -= (r[i].w * r[j].w) * rsqrt(r2);
        }

        double vx = v[i].x * v[i].x;
        double vy = v[i].y * v[i].y;
        double vz = v[i].z * v[i].z;
        double v2 = vx + vy + vz;

        ekin_tmp = 0.5 * r[i].w * v2;

        ekin[i] = ekin_tmp;
        epot[i] = epot_tmp;
    }
}

void Hermite4GPU::get_kernel_error(){
    #ifdef KERNEL_ERROR_DEBUG
        std::cerr << "[Error] : ";
        std::cerr << hipGetErrorString(hipGetLastError()) << std::endl;
    #endif
}

void Hermite4GPU::gpu_timer_start(){
    hipEventRecord(start);
}

float Hermite4GPU::gpu_timer_stop(std::string f){
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float msec = 0;
    hipEventElapsedTime(&msec, start, stop);
    #if KERNEL_TIME
    if (f != "")
        std::cout << "[Time] " << f << " : " << msec << " msec" << std::endl;
    #endif
    return msec;
}

void Hermite4GPU::force_calculation(Predictor pi, Predictor pj, Forces &fi)
{
    double rx = pj.r[0] - pi.r[0];
    double ry = pj.r[1] - pi.r[1];
    double rz = pj.r[2] - pi.r[2];

    double vx = pj.v[0] - pi.v[0];
    double vy = pj.v[1] - pi.v[1];
    double vz = pj.v[2] - pi.v[2];

    double r2     = rx*rx + ry*ry + rz*rz + ns->e2;
    double rinv   = 1.0/sqrt(r2);
    double r2inv  = rinv  * rinv;
    double r3inv  = r2inv * rinv;
    double r5inv  = r2inv * r3inv;
    double mr3inv = r3inv * pj.m;
    double mr5inv = r5inv * pj.m;

    double rv = rx*vx + ry*vy + rz*vz;

    fi.a[0] += (rx * mr3inv);
    fi.a[1] += (ry * mr3inv);
    fi.a[2] += (rz * mr3inv);

    fi.a1[0] += (vx * mr3inv - (3 * rv ) * rx * mr5inv);
    fi.a1[1] += (vy * mr3inv - (3 * rv ) * ry * mr5inv);
    fi.a1[2] += (vz * mr3inv - (3 * rv ) * rz * mr5inv);
}

void Hermite4GPU::integration()
{
    ns->gtime.integration_ini = omp_get_wtime();

    double ATIME = 1.0e+10; // Actual integration time
    double ITIME = ns->snapshot_time;     // Integration time
    int nact     = 0;       // Active particles
    int nsteps   = 0;       // Amount of steps per particles on the system
    static long long interactions = 0;


    int max_threads = omp_get_max_threads();
    omp_set_num_threads( max_threads - 1);

    init_acc_jrk();
    init_dt(ATIME, ETA_S, ITIME);

    ns->en.ini = get_energy_gpu();   // Initial calculation of the energy of the system
    //ns->en.ini = nu->get_energy(0);   // Initial calculation of the energy of the system
    ns->en.tmp = ns->en.ini;
    std::cout << ns->en.ini << std::endl;

    //ns->hmr_time = nu->get_half_mass_relaxation_time();
    //ns->cr_time  = nu->get_crossing_time();

    logger->print_info();
    logger->write_info();
    logger->print_energy_log(ITIME, ns->iterations, interactions, nsteps, ns->en.ini);

    int snap_number = ns->snapshot_number;
    logger->write_snapshot(snap_number, ITIME);
    snap_number++;

    if (ns->ops.print_all)
    {
        logger->print_all(ITIME);
    }
    if (ns->ops.print_lagrange)
    {
        nu->lagrange_radii();
        logger->print_lagrange_radii(ITIME, nu->layers_radii);
    }

    while (ITIME < ns->integration_time)
    {
        ITIME = ATIME;

        nact = find_particles_to_move(ITIME);

        save_old_acc_jrk(nact);

        predicted_pos_vel(ITIME);

        update_acc_jrk(nact);

        correction_pos_vel(ITIME, nact);

        // Update the amount of interactions counter
        interactions += nact * ns->n;

        // Find the next integration time
        next_integration_time(ATIME);


        if(nact == ns->n)
        {
            //assert(nact == ns->n);
            logger->print_energy_log(ITIME, ns->iterations, interactions, nsteps, get_energy_gpu());
            if (ns->ops.print_all)
            {
                logger->print_all(ITIME);
            }
            if (ns->ops.print_lagrange)
            {
                nu->lagrange_radii();
                logger->print_lagrange_radii(ITIME, nu->layers_radii);
            }
            logger->write_snapshot(snap_number, ITIME);
            snap_number++;
        }

        // Update nsteps with nact
        nsteps += nact;

        // Increase iteration counter
        ns->iterations++;

    }
    ns->gtime.integration_end =  omp_get_wtime() - ns->gtime.integration_ini;
    logger->write_snapshot(snap_number, ITIME);
    //logger->add_info(std::string("SnapshotNumber:"), std::to_string(snap_number));
    logger->add_info(std::string("SnapshotNumber:"), std::string(SSTR(snap_number)));
}
