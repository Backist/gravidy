#include "hip/hip_runtime.h"
#include "dynamics_gpu_kernels.cuh"

__device__ void gpu_force_calculation(double4 i_pos, double4 i_vel,
                            double4 j_pos, double4 j_vel,
                            double4 &acc,  double4 &jrk,
                            float j_mass)
{
//    double dx =  j_pos.x - i_pos.x;
//    double dy =  j_pos.y - i_pos.y;
//    double dz =  j_pos.z - i_pos.z;
//    double dvx = j_vel.x - i_vel.x;
//    double dvy = j_vel.y - i_vel.y;
//    double dvz = j_vel.z - i_vel.z;
//    double r2 =  dx*dx + dy*dy + dz*dz;
//    double rv =  dx*dvx + dy*dvy + dz*dvz;
//    double rinv1 = rsqrtf(r2);
//    double rinv2 = rinv1 * rinv1;
//    double mrinv1 = j_mass * rinv1;
//    double mrinv3 = mrinv1 * rinv2;
//    rv *= 3.f * rinv2;
//    acc.x += mrinv3 * dx;
//    acc.y += mrinv3 * dy;
//    acc.z += mrinv3 * dz;
//    jrk.x += mrinv3 * (dvx - rv * dx);
//    jrk.y += mrinv3 * (dvy - rv * dy);
//    jrk.z += mrinv3 * (dvz - rv * dz);

     double3 rr = {j_pos.x - i_pos.x, j_pos.y - i_pos.y, j_pos.z - i_pos.z};
     double3 vv = {j_vel.x - i_vel.x, j_vel.y - i_vel.y, j_vel.z - i_vel.z};

     double r2 = rr.x*rr.x + rr.y*rr.y + rr.z*rr.z + E2;

     double rinv   = 1/sqrt(r2);
     double r2inv  = rinv  * rinv;

     double r3inv  = r2inv * rinv;
     double r5inv  = r2inv * r3inv;
     double mr3inv = r3inv * j_mass;
     double mr5inv = r5inv * j_mass;

     double rv = rr.x*vv.x + rr.y*vv.y + rr.z*vv.z;

     acc.x += (rr.x * mr3inv);
     acc.y += (rr.y * mr3inv);
     acc.z += (rr.z * mr3inv);

     jrk.x += (vv.x * mr3inv - (3 *  rv) * rr.x * mr5inv);
     jrk.y += (vv.y * mr3inv - (3 *  rv) * rr.y * mr5inv);
     jrk.z += (vv.z * mr3inv - (3 *  rv) * rr.z * mr5inv);
}


__global__ void
k_energy(double4 *r, double4 *v, double *ekin, double *epot, float *m, int n)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j;

    if (i < n)
    {
        double epot_tmp = 0.0;
        for (j = i+1; j < n; j++)
        {
            double3 rr = {r[j].x - r[i].x, r[j].y - r[i].y, r[j].z - r[i].z};
            double r2 = rr.x*rr.x + rr.y*rr.y + rr.z*rr.z;

            epot_tmp -= (m[i] * m[j]) * rsqrt(r2);
        }

        double3 vv = {v[i].x * v[i].x, v[i].y * v[i].y, v[i].z * v[i].z};

        double v2 = vv.x + vv.y + vv.z;
        double ekin_tmp = 0.5 * m[i] * v2;

        ekin[i] = ekin_tmp;
        epot[i] = epot_tmp;
    }
}

__global__ void
k_init_acc_jrk(double4 *r, double4 *v, double4 *a, double4 *j, float *m, int n)
{

    extern __shared__ double4 sh[];
    double4 *sr = (double4*)sh;
    double4 *sv = (double4*)&sr[blockDim.x];

    double4 aa = {0.0, 0.0, 0.0, 0.0};
    double4 jj = {0.0, 0.0, 0.0, 0.0};
    float mj;

    int id = threadIdx.x + blockDim.x * blockIdx.x;
    int tx = threadIdx.x;

    if (id < n)
    {
        double4 pos = r[id];
        double4 vel = v[id];

        int tile = 0;
        for (int i = 0; i < n; i += BSIZE)
        {
            int idx = tile * BSIZE + tx;

            sr[tx]   = r[idx];
            sv[tx]   = v[idx];
            mj = m[idx];
            __syncthreads();

            for (int k = 0; k < BSIZE; k++)
            {

                gpu_force_calculation(pos, vel, sr[k], sv[k], aa, jj, mj);
//                double3 rr = {sr[k].x - pos.x, sr[k].y - pos.y, sr[k].z - pos.z};
//                double3 vv = {sv[k].x - vel.x, sv[k].y - vel.y, sv[k].z - vel.z};
//
//                double r2 = rr.x*rr.x + rr.y*rr.y + rr.z*rr.z + E2;
//
//                double rinv   = rsqrt(r2);
//                double r2inv  = rinv  * rinv;
//                double r3inv  = r2inv * rinv;
//                double r5inv  = r2inv * r3inv;
//                double mr3inv = r3inv * mj;
//                double mr5inv = r5inv * mj;
//
//                double rv = rr.x*vv.x + rr.y*vv.y + rr.z*vv.z + E2;
//
//                aa.x += (rr.x * mr3inv);
//                aa.y += (rr.y * mr3inv);
//                aa.z += (rr.z * mr3inv);
//
//                jj.x += (vv.x * mr3inv - (3 *  rv) * rr.x * mr5inv);
//                jj.y += (vv.y * mr3inv - (3 *  rv) * rr.y * mr5inv);
//                jj.z += (vv.z * mr3inv - (3 *  rv) * rr.z * mr5inv);
            }
            __syncthreads();
            tile++;
        }

        a[id] = aa;
        j[id] = jj;
    }
}

__global__ void
k_update_acc_jrk_single(double4 *new_a, double4 *new_j, double4 *r, double4 *v,
                        float *m, int n, int current)
{

    double4 aa = {0.0, 0.0, 0.0, 0.0};
    double4 jj = {0.0, 0.0, 0.0, 0.0};
    double4 pos = r[current];
    double4 vel = v[current];

    int id = threadIdx.x + blockDim.x * blockIdx.x;

    if (id < n)
    {
        if(id != current)
        {
            float mj = m[id];
            double3 rr = {r[id].x - pos.x, r[id].y - pos.y, r[id].z - pos.z};
            double3 vv = {v[id].x - vel.x, v[id].y - vel.y, v[id].z - vel.z};

            double r2 = rr.x*rr.x + rr.y*rr.y + rr.z*rr.z + E2;

            //double rinv = rsqrt(r2);
            double rinv = 1/sqrt(r2);
            double r2inv = rinv  * rinv;
            double r3inv = r2inv * rinv;
            double r5inv = r2inv * r3inv;
            double mr3inv = r3inv * mj;
            double mr5inv = r5inv * mj;

            double rv = rr.x*vv.x + rr.y*vv.y + rr.z*vv.z;

            aa.x = (rr.x * mr3inv);
            aa.y = (rr.y * mr3inv);
            aa.z = (rr.z * mr3inv);

            jj.x = (vv.x * mr3inv - (3 * rv ) * rr.x * mr5inv);
            jj.y = (vv.y * mr3inv - (3 * rv ) * rr.y * mr5inv);
            jj.z = (vv.z * mr3inv - (3 * rv ) * rr.z * mr5inv);

        }

        new_a[id] = aa;
        new_j[id] = jj;
    }
}

__global__ void
k_predicted_pos_vel(double4 *d_r,   double4 *d_v,   double4 *d_a, double4 *d_j,
                    double4 *d_p_r, double4 *d_p_v, double *d_t, double ITIME, int n)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    if (i < n)
    {
        float dt = ITIME - d_t[i];
        float dt2 = (dt  * dt);
        float dt3 = (dt2 * dt);

        d_p_r[i].x = (dt3/6 * d_j[i].x) + (dt2/2 * d_a[i].x) + (dt * d_v[i].x) + d_r[i].x;
        d_p_r[i].y = (dt3/6 * d_j[i].y) + (dt2/2 * d_a[i].y) + (dt * d_v[i].y) + d_r[i].y;
        d_p_r[i].z = (dt3/6 * d_j[i].z) + (dt2/2 * d_a[i].z) + (dt * d_v[i].z) + d_r[i].z;

        d_p_v[i].x = (dt2/2 * d_j[i].x) + (dt * d_a[i].x) + d_v[i].x;
        d_p_v[i].y = (dt2/2 * d_j[i].y) + (dt * d_a[i].y) + d_v[i].y;
        d_p_v[i].z = (dt2/2 * d_j[i].z) + (dt * d_a[i].z) + d_v[i].z;
    }
}



__global__ void k_update_2d(int *move, double4 *new_acc, double4 *new_jrk,
                            double4 *r,       double4 *v,
                            float *m,         int total,  int n)
{

    int xbid  = blockIdx.x;
    int ybid  = blockIdx.y;
    int tx  = threadIdx.x;
    int gid = tx + blockDim.x * xbid;

    int j_ini = (total * ybid)/NJBLOCK;
    int j_end = (total * (ybid + 1))/NJBLOCK;

    double4 pos = r[gid];
    double4 vel = v[gid];
    double4 j_acc  = {0.0,0.0,0.0,0.0};
    double4 j_jrk = {0.0,0.0,0.0,0.0};

    for (int j = j_ini; j < j_end; j+=BSIZE)
    {
        // Shared memory of BSIZE for j-particles
        __shared__ double4 s_r[BSIZE];
        __shared__ double4 s_v[BSIZE];
        __syncthreads();

        // Load of the r and v to shared memory of the j-particle
        double4 *src_r = (double4 *)&r[j];
        double4 *src_v = (double4 *)&v[j];
        double4 *dst_r = (double4 *)s_r;
        double4 *dst_v = (double4 *)s_v;

        dst_r[tx]         = src_r[tx];
        dst_r[BSIZE + tx] = src_r[BSIZE + tx];

        dst_v[tx]         = src_v[tx];
        dst_v[BSIZE + tx] = src_v[BSIZE + tx];

        __syncthreads();

        // If we need to work with an incomplete block
        if (j_end - j < BSIZE)
        {
            #pragma unroll 4
            for (int jj = 0; jj < j_end - j; jj++)
            {
                double4 pos_j = s_r[jj];
                double4 vel_j = s_v[jj];
                gpu_force_calculation(pos, vel, pos_j, vel_j, j_acc, j_jrk, m[jj]);
            }
        }
        else
        {
            #pragma unroll 4
            for (int jj = 0; jj < BSIZE; jj++)
            {
                double4 pos_j = s_r[jj];
                double4 vel_j = s_v[jj];
                gpu_force_calculation(pos, vel, pos_j, vel_j, j_acc, j_jrk, m[jj]);
            }
        }
     }
    new_acc[gid + n * ybid]  = j_acc;
    new_jrk[gid + n * ybid] =  j_jrk;
}

/*
__global__ void k_reduce_energy(float *d_in, float *d_out, int n)
{
    extern __shared__ float data[];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (BSIZE * 2) + tid;
    unsigned int gridSize = BSIZE * 2 * gridDim.x;
    float sum = 0.0f;

    while (i < n)
    {
        sum = sum + d_in[i];
        if (i + BSIZE < n)
            sum = sum + d_in[i+BSIZE];
        i += gridSize;
    }

    data[tid] = sum;
    __syncthreads();

    if(BSIZE >= 512){if(tid < 256){data[tid] += data[tid + 256];} __syncthreads();}
    if(BSIZE >= 256){if(tid < 128){data[tid] += data[tid + 128];} __syncthreads();}
    if(BSIZE >= 128){if(tid <  64){data[tid] += data[tid +  64];} __syncthreads();}

    if(tid < 32)
    {
        volatile float* smem = data;
        if (BSIZE >= 62) { smem[tid] += smem[tid + 32]; }
        if (BSIZE >= 32) { smem[tid] += smem[tid + 16]; }
        if (BSIZE >= 16) { smem[tid] += smem[tid +  8]; }
        if (BSIZE >=  8) { smem[tid] += smem[tid +  4]; }
        if (BSIZE >=  4) { smem[tid] += smem[tid +  2]; }
        if (BSIZE >=  2) { smem[tid] += smem[tid +  1]; }
    }

    if(tid == 0) d_out[blockIdx.x] = data[0];
}
*/

__global__ void k_reduce(double4 *d_in, double4 *d_out, int n)
{
    extern __shared__ double4 sdata[];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (BSIZE * 2) + tid;
    unsigned int gridSize = BSIZE * 2 * gridDim.x;
    double4 sum = {0.0,0.0,0.0,0.0};

    while (i < n)
    {
        sum = sum + d_in[i];
        if (i + BSIZE < n)
            sum = sum + d_in[i+BSIZE];
        i += gridSize;
    }

    sdata[tid] = sum;
    __syncthreads();

    if(BSIZE >= 512){if(tid < 256){sdata[tid] += sdata[tid+256];} __syncthreads();}
    if(BSIZE >= 256){if(tid < 128){sdata[tid] += sdata[tid+128];} __syncthreads();}
    if(BSIZE >= 128){if(tid <  64){sdata[tid] += sdata[tid+ 64];} __syncthreads();}


    if(tid < 32)
    {
        volatile double4* smem = sdata;
        if (BSIZE >= 62) { smem[tid] += smem[tid + 32]; }
        if (BSIZE >= 32) { smem[tid] += smem[tid + 16]; }
        if (BSIZE >= 16) { smem[tid] += smem[tid +  8]; }
        if (BSIZE >=  8) { smem[tid] += smem[tid +  4]; }
        if (BSIZE >=  4) { smem[tid] += smem[tid +  2]; }
        if (BSIZE >=  2) { smem[tid] += smem[tid +  1]; }
    }

    if(tid == 0) d_out[blockIdx.x] = sdata[0];
}


/*
__global__ void k_update_acc_jrk(double4 *r, double4 *v, double4 *a, double4 *j,
                                 float *m,   int *move,  int n,      int total)
{

    extern __shared__ double4 sh[];
    double4 *s_r = (double4*)sh;
    double4 *s_v = (double4*)&s_r[blockDim.x];

    double4 pos = {0.0, 0.0, 0.0, 0.0};
    double4 vel = {0.0, 0.0, 0.0, 0.0};
    double4 tmp_a = {0.0, 0.0, 0.0, 0.0};
    double4 tmp_j = {0.0, 0.0, 0.0, 0.0};

    int ii = threadIdx.x + blockDim.x * blockIdx.x;
    int tx = threadIdx.x;
    int id = move[ii];

    if (id != -1)
    {
        pos   = r[id];
        vel   = v[id];
    }

    for (int i = 0; i < n; i += BSIZE)
    {
        int idx = i + tx;

        s_r[tx]  = r[idx];
        s_v[tx]  = v[idx];
        float mj = m[idx];
        __syncthreads();

        for (int k = 0; k < BSIZE; k++)
        {
            if(id != idx)
            {
                double3 tmp_r = { s_r[k].x - pos.x, s_r[k].y - pos.y, s_r[k].z - pos.z};
                double3 tmp_v = { s_v[k].x - vel.x, s_v[k].y - vel.y, s_v[k].z - vel.z};

                double r2 = tmp_r.x*tmp_r.x + tmp_r.y*tmp_r.y + tmp_r.z*tmp_r.z + E2;

                double rinv = rsqrt(r2);
                double r2inv = rinv  * rinv;
                double r3inv = r2inv * rinv;
                double r5inv = r2inv * r3inv;
                double mr3inv = r3inv * mj;
                double mr5inv = r5inv * mj;

                tmp_a.x += tmp_r.x * mr3inv;
                tmp_a.y += tmp_r.y * mr3inv;
                tmp_a.z += tmp_r.z * mr3inv;

                tmp_j.x += tmp_v.x * mr3inv + (3 * tmp_v.x * tmp_r.x * tmp_r.x) * mr5inv;
                tmp_j.y += tmp_v.y * mr3inv + (3 * tmp_v.y * tmp_r.y * tmp_r.y) * mr5inv;
                tmp_j.z += tmp_v.z * mr3inv + (3 * tmp_v.z * tmp_r.z * tmp_r.z) * mr5inv;
            }
        }
        __syncthreads();
    }

    if (id != -1)
    {
        a[id] = tmp_a;
        j[id] = tmp_j;
    }
}
*/


__global__ void
k_update_acc_jrk_simple(double4 *r, double4 *v, double4 *a, double4 *j, float *m, int *move, int n, int total)
{
    extern __shared__ double4 sh[];
    double4 *sr = (double4*)sh;
    double4 *sv = (double4*)&sr[blockDim.x];

    double4 aa = {0.0, 0.0, 0.0, 0.0};
    double4 jj = {0.0, 0.0, 0.0, 0.0};
    double4 pos = {0.0, 0.0, 0.0, 0.0};
    double4 vel = {0.0, 0.0, 0.0, 0.0};

    float mj;

    int id = threadIdx.x + blockDim.x * blockIdx.x;
    int tx = threadIdx.x;


    int id_move = move[(int)(id/BSIZE)];

    if(id_move != -1)
    {
        pos = r[id_move];
        vel = v[id_move];
    }


    int tile = 0;
    for (int i = 0; i < n; i += BSIZE)
    {
        int idx = tile * BSIZE + tx;

        sr[tx]   = r[idx];
        sv[tx]   = v[idx];
        mj = m[idx];
        __syncthreads();

        for (int k = 0; k < BSIZE; k++)
        {
            gpu_force_calculation(pos, vel, sr[k], sv[k], aa, jj, mj);
        }
        __syncthreads();
        tile++;
    }

        a[id] = aa;
        j[id] = jj;

}
