#include "hip/hip_runtime.h"
#include "dynamics_gpu_kernels.cuh"

__global__ void
k_energy(double4 *r, double4 *v, double *ekin, double *epot, float *m, int n)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j;

    if (i < n)
    {
        double epot_tmp = 0.0;
        for (j = i+1; j < n; j++)
        {
            double3 rr = {r[j].x - r[i].x, r[j].y - r[i].y, r[j].z - r[i].z};
            double r2 = rr.x*rr.x + rr.y*rr.y + rr.z*rr.z;

            epot_tmp -= (m[i] * m[j]) * rsqrt(r2);
        }

        double3 vv = {v[i].x * v[i].x, v[i].y * v[i].y, v[i].z * v[i].z};

        double v2 = vv.x + vv.y + vv.z;
        double ekin_tmp = 0.5 * m[i] * v2;

        ekin[i] = ekin_tmp;
        epot[i] = epot_tmp;
    }
}

__global__ void k_update_acc_jrk_single
(double4 *new_a, double4 *new_j, double4 *r, double4 *v, float *m, int n, int current)
{

    double4 aa = {0.0, 0.0, 0.0, 0.0};
    double4 jj = {0.0, 0.0, 0.0, 0.0};
    double4 pos = r[current];
    double4 vel = v[current];

    int id = threadIdx.x + blockDim.x * blockIdx.x;

    if (id < n)
    {
        if(id != current)
        {
            gpu_force_calculation(pos, vel, r[id], v[id], aa, jj, m[id]);
        }

        new_a[id] = aa;
        new_j[id] = jj;
    }
}

__global__ void k_update_2d(int *move,  double4 *new_acc, double4 *new_jrk,
                            double4 *r, double4 *v,       float *m,
                            int total,  int n)
{

    int xbid  = blockIdx.x;
    int ybid  = blockIdx.y;
    int tx  = threadIdx.x;
    int gid = tx + blockDim.x * xbid;

    int j_ini = (total * ybid)/NJBLOCK;
    int j_end = (total * (ybid + 1))/NJBLOCK;

    double4 pos = r[gid];
    double4 vel = v[gid];
    double4 j_acc  = {0.0,0.0,0.0,0.0};
    double4 j_jrk = {0.0,0.0,0.0,0.0};

    for (int j = j_ini; j < j_end; j+=BSIZE)
    {
        // Shared memory of BSIZE for j-particles
        __shared__ double4 s_r[BSIZE];
        __shared__ double4 s_v[BSIZE];
        __syncthreads();

        // Load of the r and v to shared memory of the j-particle
        double4 *src_r = (double4 *)&r[j];
        double4 *src_v = (double4 *)&v[j];
        double4 *dst_r = (double4 *)s_r;
        double4 *dst_v = (double4 *)s_v;

        dst_r[tx]         = src_r[tx];
        dst_r[BSIZE + tx] = src_r[BSIZE + tx];

        dst_v[tx]         = src_v[tx];
        dst_v[BSIZE + tx] = src_v[BSIZE + tx];

        __syncthreads();

        // If we need to work with an incomplete block
        if (j_end - j < BSIZE)
        {
            #pragma unroll 4
            for (int jj = 0; jj < j_end - j; jj++)
            {
                double4 pos_j = s_r[jj];
                double4 vel_j = s_v[jj];
                gpu_force_calculation(pos, vel, pos_j, vel_j, j_acc, j_jrk, m[jj]);
            }
        }
        else
        {
            #pragma unroll 4
            for (int jj = 0; jj < BSIZE; jj++)
            {
                double4 pos_j = s_r[jj];
                double4 vel_j = s_v[jj];
                gpu_force_calculation(pos, vel, pos_j, vel_j, j_acc, j_jrk, m[jj]);
            }
        }
     }
    new_acc[gid + n * ybid]  = j_acc;
    new_jrk[gid + n * ybid] =  j_jrk;
}

__global__ void k_reduce(double4 *d_in, double4 *d_out, int n)
{
    extern __shared__ double4 sdata[];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (BSIZE * 2) + tid;
    unsigned int gridSize = BSIZE * 2 * gridDim.x;
    double4 sum = {0.0,0.0,0.0,0.0};

    while (i < n)
    {
        sum = sum + d_in[i];
        if (i + BSIZE < n)
            sum = sum + d_in[i+BSIZE];
        i += gridSize;
    }

    sdata[tid] = sum;
    __syncthreads();

    if(BSIZE >= 512){if(tid < 256){sdata[tid] += sdata[tid+256];} __syncthreads();}
    if(BSIZE >= 256){if(tid < 128){sdata[tid] += sdata[tid+128];} __syncthreads();}
    if(BSIZE >= 128){if(tid <  64){sdata[tid] += sdata[tid+ 64];} __syncthreads();}


    if(tid < 32)
    {
        volatile double4* smem = sdata;
        if (BSIZE >= 62) { smem[tid] += smem[tid + 32]; }
        if (BSIZE >= 32) { smem[tid] += smem[tid + 16]; }
        if (BSIZE >= 16) { smem[tid] += smem[tid +  8]; }
        if (BSIZE >=  8) { smem[tid] += smem[tid +  4]; }
        if (BSIZE >=  4) { smem[tid] += smem[tid +  2]; }
        if (BSIZE >=  2) { smem[tid] += smem[tid +  1]; }
    }

    if(tid == 0) d_out[blockIdx.x] = sdata[0];
}

__global__ void k_update_acc_jrk_simple
(double4 *r, double4 *v, double4 *a, double4 *j, float *m, int *move, int n, int total)
{
    extern __shared__ double4 sh[];
    double4 *sr = (double4*)sh;
    double4 *sv = (double4*)&sr[blockDim.x];

    double4 aa  = {0.0, 0.0, 0.0, 0.0};
    double4 jj  = {0.0, 0.0, 0.0, 0.0};
    double4 pos = {0.0, 0.0, 0.0, 0.0};
    double4 vel = {0.0, 0.0, 0.0, 0.0};
    float mj;

    int id = threadIdx.x + blockDim.x * blockIdx.x;
    int tx = threadIdx.x;

    int id_move = move[(int)(id/BSIZE)];

    if(id_move != -1)
    {
        pos = r[id_move];
        vel = v[id_move];
    }

    int tile = 0;
    for (int i = 0; i < n; i += BSIZE)
    {
        int idx = tile * BSIZE + tx;

        sr[tx]   = r[idx];
        sv[tx]   = v[idx];
        mj = m[idx];
        __syncthreads();

        for (int k = 0; k < BSIZE; k++)
        {
            if(id_move != -1)
                gpu_force_calculation(pos, vel, sr[k], sv[k], aa, jj, mj);
        }
        __syncthreads();
        tile++;
    }

    if(id_move != -1)
    {
        a[id] = aa;
        j[id] = jj;
    }
}

/*
 * @fn k_init_acc_jrk
 *
 * @param to do
 *
 * @desc GPU Kernel to perform the calculation of the initial acceleration
 *       and jerk of the system.
 *
 * @note Working properly
 */
__global__ void k_init_acc_jrk
(double4 *r, double4 *v, double4 *a, double4 *j, float *m, int n)
{

    extern __shared__ double4 sh[];
    double4 *sr = (double4*)sh;
    double4 *sv = (double4*)&sr[blockDim.x];

    double4 aa = {0.0, 0.0, 0.0, 0.0};
    double4 jj = {0.0, 0.0, 0.0, 0.0};
    float mj;

    int id = threadIdx.x + blockDim.x * blockIdx.x;
    int tx = threadIdx.x;

    if (id < n)
    {
        double4 pos = r[id];
        double4 vel = v[id];

        int tile = 0;
        for (int i = 0; i < n; i += BSIZE)
        {
            int idx = tile * BSIZE + tx;

            sr[tx]   = r[idx];
            sv[tx]   = v[idx];
            mj = m[idx];
            __syncthreads();

            for (int k = 0; k < BSIZE; k++)
            {
                gpu_force_calculation(pos, vel, sr[k], sv[k], aa, jj, mj);
            }
            __syncthreads();
            tile++;
        }

        a[id] = aa;
        j[id] = jj;
    }
}

/*
 * @fn k_force_calculation
 *
 * @param to do
 *
 * @desc GPU Kernel which calculates the interaction between
 *       a i-particle and a j-particle.
 *
 * @note Working properly.
 *
 */
__device__ void k_force_calculation(double4 i_pos, double4 i_vel,
                                    double4 j_pos, double4 j_vel,
                                    double4 &acc,  double4 &jrk,
                                    float   j_mass)
{
//    double mrinv1 = j_mass * rinv1;
//    double mrinv3 = mrinv1 * rinv2;
//    rv *= 3.f * rinv2;
//    acc.x += mrinv3 * dx;
//    acc.y += mrinv3 * dy;
//    acc.z += mrinv3 * dz;
//    jrk.x += mrinv3 * (dvx - rv * dx);
//    jrk.y += mrinv3 * (dvy - rv * dy);
//    jrk.z += mrinv3 * (dvz - rv * dz);

     double3 rr = {j_pos.x - i_pos.x, j_pos.y - i_pos.y, j_pos.z - i_pos.z};
     double3 vv = {j_vel.x - i_vel.x, j_vel.y - i_vel.y, j_vel.z - i_vel.z};
     double r2  = (rr.x * rr.x) + (rr.y * rr.y) + (rr.z * rr.z) + E2;
     double rv  = (rr.x * vv.x) + (rr.y * vv.y) + (rr.z * vv.z);

     double rinv   = 1/sqrt(r2);
     double r2inv  = rinv  * rinv;
     double r3inv  = r2inv * rinv;
     double r5inv  = r2inv * r3inv;
     double mr3inv = r3inv * j_mass;
     double mr5inv = r5inv * j_mass;

     acc.x += (rr.x * mr3inv);
     acc.y += (rr.y * mr3inv);
     acc.z += (rr.z * mr3inv);

     jrk.x += (vv.x * mr3inv - (3 * rv) * rr.x * mr5inv);
     jrk.y += (vv.y * mr3inv - (3 * rv) * rr.y * mr5inv);
     jrk.z += (vv.z * mr3inv - (3 * rv) * rr.z * mr5inv);
}

/*
 * @fn k_predicted_pos_vel
 *
 * @param to do
 *
 * @desc GPU Kernel to calculate the predicted position and velocity
 *       of all the particles.
 *
 * @note Working properly.
 */
__global__ void
k_predicted_pos_vel(double4 *d_r,   double4 *d_v,   double4 *d_a, double4 *d_j,
                    double4 *d_p_r, double4 *d_p_v, double *d_t, double ITIME, int n)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    if (i < n)
    {
        float dt = ITIME - d_t[i];
        float dt2 = (dt  * dt)/2;
        float dt3 = (dt2 * dt)/6;

        d_p_r[i].x = (dt3 * d_j[i].x) + (dt2 * d_a[i].x) + (dt * d_v[i].x) + d_r[i].x;
        d_p_r[i].y = (dt3 * d_j[i].y) + (dt2 * d_a[i].y) + (dt * d_v[i].y) + d_r[i].y;
        d_p_r[i].z = (dt3 * d_j[i].z) + (dt2 * d_a[i].z) + (dt * d_v[i].z) + d_r[i].z;

        d_p_v[i].x = (dt2 * d_j[i].x) + (dt  * d_a[i].x) + d_v[i].x;
        d_p_v[i].y = (dt2 * d_j[i].y) + (dt  * d_a[i].y) + d_v[i].y;
        d_p_v[i].z = (dt2 * d_j[i].z) + (dt  * d_a[i].z) + d_v[i].z;
    }
}
