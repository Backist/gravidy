#include "hip/hip_runtime.h"
#include "dynamics_gpu.cuh"

struct sum_double4
{
    __host__ __device__
    double4 operator() (const double4 l, const double4 r)
    {
        return make_double4(l.x + r.x, l.y + r.y, l.z + r.z, 0.0);
    }
};

/*
 * @fn gpu_energy()
 *
 * @brief
 *  Host function which call the kernel to calculate
 *  the energy of the system.
 */
__host__
double
gpu_energy(bool type)
{
    int d4_size = sizeof(double4) * n;
    int f1_size = sizeof(float)  * n;

    int nthreads = BSIZE;
    int nblocks  = std::ceil(n/(float)nthreads);

    if(type)
    {
        hipMemcpy(d_p_r, h_p_r, d4_size, hipMemcpyHostToDevice);
        hipMemcpy(d_p_v, h_p_v, d4_size, hipMemcpyHostToDevice);
        k_energy <<< nblocks, nthreads >>> (d_p_r, d_p_v, d_ekin, d_epot, d_m, n);
    }
    else if(!type)
    {
        hipMemcpy(d_r, h_r, d4_size, hipMemcpyHostToDevice);
        hipMemcpy(d_v, h_v, d4_size, hipMemcpyHostToDevice);
        k_energy <<< nblocks, nthreads >>> (d_r, d_v, d_ekin, d_epot, d_m, n);
    }

    hipDeviceSynchronize();
//    std::cerr << hipGetErrorString(hipGetLastError()) << std::endl;

    hipMemcpy(h_ekin, d_ekin, f1_size, hipMemcpyDeviceToHost);
    hipMemcpy(h_epot, d_epot, f1_size, hipMemcpyDeviceToHost);

    ekin = 0;
    epot = 0;
    for (int i = 0; i < n; i++)
    {
        ekin += h_ekin[i];
        epot += h_epot[i];
    }

    return ekin + epot;

}

/*
 * @fn gpu_init_acc_jerk()
 *
 * @brief
 *  Host function which call the kernel to calculate
 *  the initial acceleration and jerk.
 */
__host__
void
gpu_init_acc_jerk()
{
    int d4_size = sizeof(double4) * n;
    int nthreads = BSIZE;
    int nblocks  = std::ceil(n/(float)nthreads);
    int smem     = BSIZE * 2 * sizeof(double4);

    hipMemcpy(d_r, h_r, d4_size, hipMemcpyHostToDevice);
    hipMemcpy(d_v, h_v, d4_size, hipMemcpyHostToDevice);

    //k_init_acc_jerk <<< nblocks, nthreads >>> (d_r, d_v, d_a, d_j, d_m, n);
    k_init_acc_jerk_tile <<< nblocks, nthreads, smem >>> (d_r, d_v, d_a, d_j, d_m, n);
    hipDeviceSynchronize();

    hipMemcpy(h_a, d_a, d4_size, hipMemcpyDeviceToHost);
    hipMemcpy(h_j, d_j, d4_size, hipMemcpyDeviceToHost);
}

/*
 * @fn gpu_update_acc_jerk()
 *
 * @param total amount of particles to update
 *
 * @brief
 *  Host function which call the kernel to calculate
 *  the new acceleration and jerk.
 */
__host__
void
gpu_update_acc_jerk_simple(int total)
{
    int d4_size = sizeof(double4) * n;
    int i1_size = sizeof(int) * n;
    int nthreads = BSIZE;
    int nblocks  = std::ceil(n/(float)nthreads);

    hipMemcpy(d_p_r,  h_p_r,  d4_size, hipMemcpyHostToDevice);
    hipMemcpy(d_p_v,  h_p_v,  d4_size, hipMemcpyHostToDevice);
    hipMemcpy(d_move, h_move, i1_size, hipMemcpyHostToDevice);

    k_update_acc_jerk_simple <<< nblocks, nthreads      >>> (d_p_r, d_p_v, d_a, d_j, d_m,
                                                             d_move, n, total);
    hipDeviceSynchronize();

    hipMemcpy(h_a, d_a, d4_size, hipMemcpyDeviceToHost);
    hipMemcpy(h_j, d_j, d4_size, hipMemcpyDeviceToHost);
}

__host__
void
gpu_update_acc_jerk_single(int total)
{
    int d4_size = sizeof(double4) * n;
    int nthreads = BSIZE;
    int nblocks  = std::ceil(n/(float)nthreads);
    int smem     = BSIZE * 2 * sizeof(double4);
    double4 new_a = {0.0,0.0,0.0,0.0};
    double4 new_j = {0.0,0.0,0.0,0.0};
    thrust::device_ptr<double4> dptr_a(d_new_a);
    thrust::device_ptr<double4> dptr_j(d_new_j);

    hipMemcpy(d_p_r,    h_p_r,    d4_size, hipMemcpyHostToDevice);
    hipMemcpy(d_p_v,    h_p_v,    d4_size, hipMemcpyHostToDevice);

    for (int k = 0; k < total; k++) {
        int i = h_move[k];
        k_update_acc_jerk_single <<< nblocks, nthreads, smem  >>> (h_p_r[i], h_p_v[i], d_new_a, d_new_j, d_p_r, d_p_v, d_m, n, i);
        
        hipDeviceSynchronize();

        try
        {
          new_a = thrust::reduce(dptr_a, dptr_a + n, make_double4(0.0,0.0,0.0,0.0), sum_double4());
          new_j = thrust::reduce(dptr_j, dptr_j + n, make_double4(0.0,0.0,0.0,0.0), sum_double4());
        }
        catch(std::bad_alloc &e)
        {
            std::cerr << "Ran out of memory while sorting" << std::endl;
            exit(-1);
        }
        catch(thrust::system_error &e)
        {
            std::cerr << "Some other error happened during sort: " << e.what() << std::endl;
            exit(-1);
        }

        h_a[i] = new_a;
        h_j[i] = new_j;
    }
}

__host__
void
gpu_update_acc_jerk_tile(int total)
{
    int d4_size = sizeof(double4) * n;
    int i1_size = sizeof(int) * n;
    int nthreads = BSIZE;
    int nblocks  = std::ceil(n/(float)nthreads);
    int smem     = BSIZE * 2 * sizeof(double4);

    hipMemcpy(d_p_r,  h_p_r,  d4_size, hipMemcpyHostToDevice);
    hipMemcpy(d_p_v,  h_p_v,  d4_size, hipMemcpyHostToDevice);
    hipMemcpy(d_move, h_move, i1_size, hipMemcpyHostToDevice);

    k_update_acc_jerk_tile <<< nblocks, nthreads, smem >>> (d_p_r, d_p_v, d_a, d_j, d_m,
                                                            d_move, n, total);
    hipDeviceSynchronize();

    hipMemcpy(h_a, d_a, d4_size, hipMemcpyDeviceToHost);
    hipMemcpy(h_j, d_j, d4_size, hipMemcpyDeviceToHost);
}

/*
 * @fn gpu_update_acc_jerk()
 *
 * @param total amount of particles to update
 *
 * @brief
 *  Host function which call the kernel to calculate
 *  the new acceleration and jerk.
 */
__host__
void
gpu_correction_pos_vel(double ITIME, int total)
{
    int d4_size = sizeof(double4) * n;
    int f1_size = sizeof(double) * n;
    int i1_size = sizeof(int) * total;
    // TO DO
    // Fix to work only with the particles who need to be moved
    int nthreads = BSIZE;
    int nblocks  = std::ceil(n/(float)nthreads);

    hipMemcpy(d_r,     h_r,     d4_size, hipMemcpyHostToDevice);
    hipMemcpy(d_v,     h_v,     d4_size, hipMemcpyHostToDevice);
    hipMemcpy(d_a,     h_a,     d4_size, hipMemcpyHostToDevice);
    hipMemcpy(d_j,     h_j,     d4_size, hipMemcpyHostToDevice);
    hipMemcpy(d_old_a, h_old_a, d4_size, hipMemcpyHostToDevice);
    hipMemcpy(d_old_j, h_old_j, d4_size, hipMemcpyHostToDevice);
    hipMemcpy(d_p_r,   h_p_r,   d4_size, hipMemcpyHostToDevice);
    hipMemcpy(d_p_v,   h_p_v,   d4_size, hipMemcpyHostToDevice);
    hipMemcpy(d_t,     h_t,     f1_size, hipMemcpyHostToDevice);
    hipMemcpy(d_dt,    h_dt,    f1_size, hipMemcpyHostToDevice);
    hipMemcpy(d_move,  h_move,  i1_size, hipMemcpyHostToDevice);

    k_correction_pos_vel<<< nblocks, nthreads >>> (d_r,     d_v,     d_a,     d_j,
                                                   d_old_a, d_old_j, 
                                                   d_p_r,   d_p_v,   d_t,     d_dt,
                                                   ITIME,   d_move,  total);
    hipDeviceSynchronize();

    hipMemcpy(h_r,  d_r,  d4_size, hipMemcpyDeviceToHost);
    hipMemcpy(h_v,  d_v,  d4_size, hipMemcpyDeviceToHost);
    hipMemcpy(h_t,  d_t,  f1_size, hipMemcpyDeviceToHost);
    hipMemcpy(h_dt, d_dt, f1_size, hipMemcpyDeviceToHost);
}
