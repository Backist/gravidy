#include "memory_gpu.cuh"

/*
 * @fn alloc_vectors_gpu()
 *
 */
void alloc_vectors_gpu()
{
    /*
     * GPU pointers
     */
    CUDA_SAFE_CALL(hipMalloc((void**)&d_r,     d4_size));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_v,     d4_size));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_a,     d4_size));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_a1,    d4_size));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_p_r,   d4_size));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_p_v,   d4_size));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_ekin,  d1_size));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_epot,  d1_size));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_t,     d1_size));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_dt,    d1_size));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_m,     f1_size));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_move,  i1_size));

    /*
     * Memset
     */
    CUDA_SAFE_CALL(hipMemset(d_r, 0, d4_size));
    CUDA_SAFE_CALL(hipMemset(d_v, 0, d4_size));
    CUDA_SAFE_CALL(hipMemset(d_a, 0, d4_size));
    CUDA_SAFE_CALL(hipMemset(d_a1,0, d4_size));
    CUDA_SAFE_CALL(hipMemset(d_p_r, 0, d4_size));
    CUDA_SAFE_CALL(hipMemset(d_p_v, 0, d4_size));
    CUDA_SAFE_CALL(hipMemset(d_ekin,  0,d1_size));
    CUDA_SAFE_CALL(hipMemset(d_epot,  0,d1_size));
    CUDA_SAFE_CALL(hipMemset(d_t,     0,d1_size));
    CUDA_SAFE_CALL(hipMemset(d_dt,    0,d1_size));
    CUDA_SAFE_CALL(hipMemset(d_m,     0,f1_size));
    CUDA_SAFE_CALL(hipMemset(d_move,  0,i1_size));
}


/*
 * @fn free_vectors_gpu()
 *
 * @brief
 *  Free memory on the GPU
 */
void free_vectors_gpu()
{
    CUDA_SAFE_CALL(hipFree(d_r));
    CUDA_SAFE_CALL(hipFree(d_v));
    CUDA_SAFE_CALL(hipFree(d_a));
    CUDA_SAFE_CALL(hipFree(d_a1));
    CUDA_SAFE_CALL(hipFree(d_m));
    CUDA_SAFE_CALL(hipFree(d_t));
    CUDA_SAFE_CALL(hipFree(d_p_r));
    CUDA_SAFE_CALL(hipFree(d_p_v));
    CUDA_SAFE_CALL(hipFree(d_dt));
    CUDA_SAFE_CALL(hipFree(d_ekin));
    CUDA_SAFE_CALL(hipFree(d_epot));
    CUDA_SAFE_CALL(hipFree(d_move));
}
