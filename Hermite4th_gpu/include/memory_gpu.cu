#include "memory_gpu.cuh"

/*
 * @fn alloc_vectors_gpu()
 *
 */
void alloc_vectors_gpu()
{
    /*
     * GPU pointers
     */
    CUDA_SAFE_CALL(hipMalloc((void**)&d_r,     d4_size));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_v,     d4_size));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_a,     d4_size));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_a1,    d4_size));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_p_r,   d4_size));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_p_v,   d4_size));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_ekin,  d1_size));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_epot,  d1_size));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_t,     d1_size));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_dt,    d1_size));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_m,     f1_size));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_move,  i1_size));
}


/*
 * @fn free_vectors_gpu()
 *
 * @brief
 *  Free memory on the GPU
 */
void free_vectors_gpu()
{
    CUDA_SAFE_CALL(hipFree(d_r));
    CUDA_SAFE_CALL(hipFree(d_v));
    CUDA_SAFE_CALL(hipFree(d_a));
    CUDA_SAFE_CALL(hipFree(d_a1));
    CUDA_SAFE_CALL(hipFree(d_m));
    CUDA_SAFE_CALL(hipFree(d_t));
    CUDA_SAFE_CALL(hipFree(d_p_r));
    CUDA_SAFE_CALL(hipFree(d_p_v));
    CUDA_SAFE_CALL(hipFree(d_dt));
    CUDA_SAFE_CALL(hipFree(d_ekin));
    CUDA_SAFE_CALL(hipFree(d_epot));
    CUDA_SAFE_CALL(hipFree(d_move));
}
